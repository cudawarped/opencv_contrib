#include "hip/hip_runtime.h"
// This file is part of OpenCV project.
// It is subject to the license terms in the LICENSE file found in the top-level directory
// of this distribution and at http://opencv.org/license.html.

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/emulation.hpp"
#include "opencv2/core/cuda/transform.hpp"
#include "opencv2/core/cuda/functional.hpp"
#include "opencv2/core/cuda/utility.hpp"
#include "opencv2/core/cuda.hpp"
#include <opencv2/core/cuda_stream_accessor.hpp>

using namespace cv::cuda;
using namespace cv::cuda::device;


namespace cv { namespace cuda { namespace device { namespace imgproc {

constexpr int blockSizeX = 32;
constexpr int blockSizeY = 16;
constexpr int momentsSize = sizeof(cv::Moments) / sizeof(double);

constexpr int m00 = offsetof(cv::Moments, m00) / sizeof(double);
constexpr int m10 = offsetof(cv::Moments, m10) / sizeof(double);
constexpr int m01 = offsetof(cv::Moments, m01) / sizeof(double);
constexpr int m20 = offsetof(cv::Moments, m20) / sizeof(double);
constexpr int m11 = offsetof(cv::Moments, m11) / sizeof(double);
constexpr int m02 = offsetof(cv::Moments, m02) / sizeof(double);
constexpr int m30 = offsetof(cv::Moments, m30) / sizeof(double);
constexpr int m21 = offsetof(cv::Moments, m21) / sizeof(double);
constexpr int m12 = offsetof(cv::Moments, m12) / sizeof(double);
constexpr int m03 = offsetof(cv::Moments, m03) / sizeof(double);

constexpr int mu20 = offsetof(cv::Moments, mu20) / sizeof(double);
constexpr int mu11 = offsetof(cv::Moments, mu11) / sizeof(double);
constexpr int mu02 = offsetof(cv::Moments, mu02) / sizeof(double);
constexpr int mu30 = offsetof(cv::Moments, mu30) / sizeof(double);
constexpr int mu21 = offsetof(cv::Moments, mu21) / sizeof(double);
constexpr int mu12 = offsetof(cv::Moments, mu12) / sizeof(double);
constexpr int mu03 = offsetof(cv::Moments, mu03) / sizeof(double);

__global__ void ComputeSpatialMoments(const cuda::PtrStepSzb img, bool binary, double* moments) {
    const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (y < img.rows && x < img.cols) {
      const unsigned int img_index = y * img.step + x;
      const unsigned char val = (!binary || img.data[img_index] == 0) ? img.data[img_index] : 1;
      if (val > 0) {
        const unsigned long x2 = x * x, x3 = x2 * x;
        const unsigned long y2 = y * y, y3 = y2 * y;

        atomicAdd(&moments[m00],           val);
        atomicAdd(&moments[m10], x       * val);
        atomicAdd(&moments[m01],      y  * val);
        atomicAdd(&moments[m20], x2      * val);
        atomicAdd(&moments[m11], x  * y  * val);
        atomicAdd(&moments[m02],      y2 * val);
        atomicAdd(&moments[m30], x3      * val);
        atomicAdd(&moments[m21], x2 * y  * val);
        atomicAdd(&moments[m12], x  * y2 * val);
        atomicAdd(&moments[m03],      y3 * val);
      }
    }
}

__global__ void ComputeSpatialMoments(const cuda::PtrStepSzb img, bool binary, float* moments) {
    const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (y < img.rows && x < img.cols) {
        const unsigned int img_index = y * img.step + x;
        const unsigned char val = (!binary || img.data[img_index] == 0) ? img.data[img_index] : 1;
        if (val > 0) {
            const unsigned long x2 = x * x, x3 = x2 * x;
            const unsigned long y2 = y * y, y3 = y2 * y;

            atomicAdd(&moments[m00], val);
            atomicAdd(&moments[m10], x * val);
            atomicAdd(&moments[m01], y * val);
            atomicAdd(&moments[m20], x2 * val);
            atomicAdd(&moments[m11], x * y * val);
            atomicAdd(&moments[m02], y2 * val);
            atomicAdd(&moments[m30], x3 * val);
            atomicAdd(&moments[m21], x2 * y * val);
            atomicAdd(&moments[m12], x * y2 * val);
            atomicAdd(&moments[m03], y3 * val);
        }
    }
}

template <typename T>
__device__ __forceinline__ T warpButterflyReduce(T value) {
    for (int i = 16; i >= 1; i /= 2)
        value += __shfl_xor_sync(0xffffffff, value, i, 32);
    return value;
}

template <typename T>
__device__ __forceinline__ T halfWarpButterflyReduce(T value) {
    for (int i = 8; i >= 1; i /= 2)
        value += __shfl_xor_sync(0xffff, value, i, 32);
    return value;
}

template <typename T>
__global__ void ComputeSpatialMomentsSharedFullReduction(const cuda::PtrStepSzb img, bool binary, T* moments) {
    const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    __shared__ T smem[10][32];

    if (threadIdx.x < 10)
        smem[threadIdx.x][threadIdx.y] = 0;
    __syncthreads();

    T val = 0;
    if (y < img.rows && x < img.cols) {
        const unsigned int img_index = y * img.step + x;
        val = (!binary || img.data[img_index] == 0) ? img.data[img_index] : 1;
    }

    const unsigned long x2 = x * x, x3 = x2 * x;
    const unsigned long y2 = y * y, y3 = y2 * y;
    T res = warpButterflyReduce(val);
    if (res) {
        smem[0][threadIdx.y] = res;
        smem[1][threadIdx.y] = warpButterflyReduce(x * val);
        smem[2][threadIdx.y] = y * res;
        smem[3][threadIdx.y] = warpButterflyReduce(x2 * val);
        smem[4][threadIdx.y] = warpButterflyReduce(x * y * val);
        smem[5][threadIdx.y] = y2 * res;
        smem[6][threadIdx.y] = warpButterflyReduce(x3 * val);
        smem[7][threadIdx.y] = warpButterflyReduce(x2 * y * val);
        smem[8][threadIdx.y] = warpButterflyReduce(x * y2 * val);
        smem[9][threadIdx.y] = y3 * res;
    }
    __syncthreads();

    if (threadIdx.x < blockSizeY && threadIdx.y < 10)
        smem[threadIdx.y][0] = halfWarpButterflyReduce(smem[threadIdx.y][threadIdx.x]);
    __syncthreads();

    if (threadIdx.y == 0 && threadIdx.x < 10)
        atomicAdd(&moments[threadIdx.x], smem[threadIdx.x][0]);
}

template <typename T>
__global__ void ComputeSpatialMomentsSharedFullReductionS1(const cuda::PtrStepSzb img, bool binary, T* moments) {
    const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    __shared__ T smem[16][10];

    if (threadIdx.y < 10 && threadIdx.x < 16)
        smem[threadIdx.x][threadIdx.y] = 0;
    __syncthreads();

    uchar val = 0;
    if (y < img.rows && x < img.cols) {
        const unsigned int img_index = y * img.step + x;
        val = (!binary || img.data[img_index] == 0) ? img.data[img_index] : 1;
    }

    const unsigned long x2 = x * x, x3 = x2 * x;
    const unsigned long y2 = y * y, y3 = y2 * y;
    T res = warpButterflyReduce(static_cast<T>(val));
    if (res) {
        smem[threadIdx.y][0] = res;
        smem[threadIdx.y][2] = y * res;
        smem[threadIdx.y][5] = y2 * res;
        smem[threadIdx.y][9] = y3 * res;
        //smem[threadIdx.y][0] = res;
        smem[threadIdx.y][1] = warpButterflyReduce(x * static_cast<T>(val));
        //smem[threadIdx.y][2] = y * res;
        smem[threadIdx.y][3] = warpButterflyReduce(x2 * static_cast<T>(val));
        smem[threadIdx.y][4] = warpButterflyReduce(x * y * static_cast<T>(val));
        //smem[threadIdx.y][5] = y2 * res;
        smem[threadIdx.y][6] = warpButterflyReduce(x3 * static_cast<T>(val));
        smem[threadIdx.y][7] = warpButterflyReduce(x2 * y * static_cast<T>(val));
        smem[threadIdx.y][8] = warpButterflyReduce(x * y2 * static_cast<T>(val));
        //smem[threadIdx.y][9] = y3 * res;
    }
    __syncthreads();

    if (threadIdx.x < 16 && threadIdx.y < 10)
        smem[threadIdx.y][0] = halfWarpButterflyReduce(smem[threadIdx.x][threadIdx.y]);
    __syncthreads();

    if (threadIdx.y == 0 && threadIdx.x < 10)
        atomicAdd(&moments[threadIdx.x], smem[threadIdx.x][0]);
}

template <typename T>
__global__ void ComputeSpatialMomentsSharedFullReductionS1F(const cuda::PtrStepSzb img, bool binary, T* moments) {
    const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    __shared__ T smem[16][10];

    if (threadIdx.y < 10)
        smem[threadIdx.x][threadIdx.y] = 0;
    __syncthreads();

    T val = 0;
    if (y < img.rows && x < img.cols) {
        const unsigned int img_index = y * img.step + x;
        val = (!binary || img.data[img_index] == 0) ? img.data[img_index] : 1;
    }

    const unsigned long x2 = x * x, x3 = x2 * x;
    const unsigned long y2 = y * y, y3 = y2 * y;
    T res = warpButterflyReduce(val);
    if (res) {
        if(threadIdx.x == 0)
            smem[threadIdx.y][0] = res;
        T tmp = warpButterflyReduce(x * val);
        if (threadIdx.x == 0) smem[threadIdx.y][1] = tmp;
        tmp = y * res;
        if (threadIdx.x == 0) smem[threadIdx.y][2] = tmp;
        tmp = warpButterflyReduce(x2 * val);
        if (threadIdx.x == 0) smem[threadIdx.y][3] = tmp;
        tmp = warpButterflyReduce(x * y * val);
        if (threadIdx.x == 0) smem[threadIdx.y][4] = tmp;
        tmp = y2 * res;
        if (threadIdx.x == 0) smem[threadIdx.y][5] = tmp;
        tmp = warpButterflyReduce(x3 * val);
        if (threadIdx.x == 0) smem[threadIdx.y][6] = tmp;
        tmp = warpButterflyReduce(x2 * y * val);
        if (threadIdx.x == 0) smem[threadIdx.y][7] = tmp;
        tmp = warpButterflyReduce(x * y2 * val);
        if (threadIdx.x == 0) smem[threadIdx.y][8] = tmp;
        tmp = y3 * res;
        if (threadIdx.x == 0) smem[threadIdx.y][9] = tmp;
    }
    __syncthreads();

    // blockSizeY - this has to be 16 as below we are using half warp reduce and above we have to enforce 32 for warp reduce
    if (threadIdx.x < blockSizeY && threadIdx.y < 10) {
        T tmp = halfWarpButterflyReduce(smem[threadIdx.x][threadIdx.y]);
        if(threadIdx.x == 0) smem[threadIdx.y][0] = tmp;
    }
    __syncthreads();

    if (threadIdx.y == 0 && threadIdx.x < 10)
        atomicAdd(&moments[threadIdx.x], smem[threadIdx.x][0]);
}


template <typename T>
__global__ void ComputeSpatialMomentsSharedFullReductionCoaleced(const cuda::PtrStepSzb img, bool binary, T* moments) {
    const unsigned int x = (blockIdx.x * blockDim.x + threadIdx.x) * 4;
    const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    constexpr int n = 10;
    __shared__ T smem[16][n];

    if (threadIdx.x < 16 && threadIdx.y < n)
        smem[threadIdx.x][threadIdx.y] = 0;
    __syncthreads();

    uchar val[4] = { 0 };
    if (y < img.rows && x < img.cols) {
        const unsigned int img_index = y * img.step + x;
        const unsigned int data = *((const unsigned int*)(&(img.data[img_index])));

        // could we read the unaligned head and tail here?
        // use threadIdx.x == 0 and loop?
        // would need to happen before we sum up the warp results - could work overly complicated, check benchmarks first

        // could perform the sum, would need to do every calc here first???

        // try to read into val array containing all


        // needs to be here for all threads in a warp to be utilized.
        #pragma unroll 4
        for (int i = 0; i < 4; i++) {
            const uchar el = ((data >> i * 8) & 0xFFU);
            val[i] = (!binary || el == 0) ? el : 1;
        }
    }

    const unsigned long y2 = y * y, y3 = y2 * y;
    #pragma unroll 4
    for (int i = 0; i < 4; i++) {
        const int iX = x + i;
        const unsigned long x2 = iX * iX, x3 = x2 * iX;
        //printf("%f\n", static_cast<T>(val[i]));
        T res = warpButterflyReduce(static_cast<T>(val[i]));
        if (res) {
            //printf("%f\n", res);
            smem[threadIdx.y][0] += res;
            smem[threadIdx.y][1] += warpButterflyReduce(iX * static_cast<T>(val[i]));
            smem[threadIdx.y][3] += warpButterflyReduce(x2 * static_cast<T>(val[i]));
            smem[threadIdx.y][4] += warpButterflyReduce(iX * y * static_cast<T>(val[i]));
            smem[threadIdx.y][6] += warpButterflyReduce(x3 * static_cast<T>(val[i]));
            smem[threadIdx.y][7] += warpButterflyReduce(x2 * y * static_cast<T>(val[i]));
            smem[threadIdx.y][8] += warpButterflyReduce(iX * y2 * static_cast<T>(val[i]));
        }
    }

    if (smem[threadIdx.y][0]) {
        smem[threadIdx.y][2] = y * smem[threadIdx.y][0];
        smem[threadIdx.y][5] = y2 * smem[threadIdx.y][0];
        smem[threadIdx.y][9] = y3 * smem[threadIdx.y][0];
    }

    __syncthreads();

    if (threadIdx.x < 16 && threadIdx.y < n)
        smem[threadIdx.y][0] = halfWarpButterflyReduce(smem[threadIdx.x][threadIdx.y]);
    __syncthreads();

    if (threadIdx.y == 0 && threadIdx.x < n)
        atomicAdd(&moments[threadIdx.x], smem[threadIdx.x][0]);
}

template <typename T>
__global__ void ComputeCentralMomentsSharedUchar(const cuda::PtrStepSzb img, bool binary, const T* m00, const T* m10, const T* m01, T* moments) {

    if (*m00 == 0 || *m10 == 0 || *m01 == 0)
        return;
    const T cX = *m10 / *m00;
    const T cY = *m01 / *m00;
    //if (!cX && !cY) return;

    const unsigned int x = (blockIdx.x * blockDim.x + threadIdx.x)*4;
    const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    constexpr int n = 7;
    __shared__ T smem[16][n];

    if (threadIdx.y < n && threadIdx.x < 16)
        smem[threadIdx.x][threadIdx.y] = 0;
    __syncthreads();

    uchar val[4] = { 0 };
    if (y < img.rows && x < img.cols) {
        //const unsigned int img_index = y * img.step + x;
        //val = (!binary || img.data[img_index] == 0) ? img.data[img_index] : 1;

        const unsigned int img_index = y * img.step + x;
        const unsigned int data = *((const unsigned int*)(&(img.data[img_index])));
#pragma unroll 4
        for (int i = 0; i < 4; i++) {
            const uchar el = ((data >> i * 8) & 0xFFU);
            val[i] = (!binary || el == 0) ? el : 1;
        }
    }


    const T y1 = y - cY, y2 = y1 * y1, y3 = y2 * y1;
    T resSum = 0;
#pragma unroll 4
    for (int i = 0; i < 4; i++) {
        const int iX = x + i;
        const T x1 = iX - cX, x2 = x1 * x1, x3 = x2 * x1;
        T res = warpButterflyReduce(static_cast<T>(val[i]));
        if (res) {
            resSum += res;
            smem[threadIdx.y][0] += warpButterflyReduce(x2 * static_cast<T>(val[i]));
            smem[threadIdx.y][1] += warpButterflyReduce(x1 * y1 * static_cast<T>(val[i]));
            smem[threadIdx.y][3] += warpButterflyReduce(x3 * static_cast<T>(val[i]));
            smem[threadIdx.y][4] += warpButterflyReduce(x2 * y1 * static_cast<T>(val[i]));
            smem[threadIdx.y][5] += warpButterflyReduce(x1 * y2 * static_cast<T>(val[i]));
        }
    }

    if (resSum) {
        smem[threadIdx.y][2] = y2 * resSum;
        smem[threadIdx.y][6] = y3 * resSum;
    }
    __syncthreads();

    //smem[threadIdx.y][0] = warpButterflyReduce(x2 * val);
    //smem[threadIdx.y][1] = warpButterflyReduce(x1 * y1 * val);
    //smem[threadIdx.y][2] = y2 * res;
    //smem[threadIdx.y][3] = warpButterflyReduce(x3 * val);
    //smem[threadIdx.y][4] = warpButterflyReduce(x2 * y1 * val);
    //smem[threadIdx.y][5] = warpButterflyReduce(x1 * y2 * val);
    //smem[threadIdx.y][6] = y3 * res;

    // blockSizeY - this has to be 16 as below we are using half warp reduce and above we have to enforce 32 for warp reduce
    if (threadIdx.x < blockSizeY && threadIdx.y < n)
        smem[threadIdx.y][0] = halfWarpButterflyReduce(smem[threadIdx.x][threadIdx.y]);
    __syncthreads();

    if (threadIdx.y == 0 && threadIdx.x < n)
        atomicAdd(&moments[threadIdx.x], smem[threadIdx.x][0]);
}

template <typename T>
__global__ void ComputeSpatialMomentsSharedPartialReduction(const cuda::PtrStepSzb img, bool binary, T* moments) {
    const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    __shared__ T smem[10];

    if (threadIdx.y == 0 && threadIdx.x < 10)
        smem[threadIdx.x] = 0;
    __syncthreads();

    T val = 0;
    if (y < img.rows && x < img.cols) {
        const unsigned int img_index = y * img.step + x;
        val = (!binary || img.data[img_index] == 0) ? img.data[img_index] : 1;
    }

    const T m00 = warpButterflyReduce(val);
    T m10 = 0, m01 = 0, m20 = 0, m11 = 0, m02 = 0, m30 = 0, m21 = 0, m12 = 0, m03 = 0;
    const unsigned long x2 = x * x, x3 = x2 * x;
    const unsigned long y2 = y * y, y3 = y2 * y;
    if (m00) {
        m10 = warpButterflyReduce(x * val);
        m01 = y * m00;
        m20 = warpButterflyReduce(x2 * val);
        m11 = warpButterflyReduce(x * y * val);
        m02 = y2 * m00;
        m30 = warpButterflyReduce(x3 * val);
        m21 = warpButterflyReduce(x2 * y * val);
        m12 = warpButterflyReduce(x * y2 * val);
        m03 = y3 * m00;
    }

    if (threadIdx.x == 0) {
        atomicAdd(&smem[0], m00);
        atomicAdd(&smem[1], m10);
        atomicAdd(&smem[2], m01);
        atomicAdd(&smem[3], m20);
        atomicAdd(&smem[4], m11);
        atomicAdd(&smem[5], m02);
        atomicAdd(&smem[6], m30);
        atomicAdd(&smem[7], m21);
        atomicAdd(&smem[8], m12);
        atomicAdd(&smem[9], m03);
    }
    __syncthreads();


    if (threadIdx.y == 0 && threadIdx.x < 10)
        atomicAdd(&moments[threadIdx.x], smem[threadIdx.x]);
}

template <typename T>
__global__ void ComputeCentralMomentsShared1(const cuda::PtrStepSzb img, bool binary, T* centroid, T* moments) {


    //if (*m00 == 0 || *m10 == 0 || *m01 == 0)
    //    return;

    const T cX = centroid[0];
    const T cY = centroid[1];

    //if (!cX && !cY) return;

    const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    //const T x1 = x - cX, y1 = y - cY;
    constexpr int n = 7;
    __shared__ T smem[16][n];

    if (threadIdx.y < n && threadIdx.x < 16)
        smem[threadIdx.x][threadIdx.y] = 0;
    __syncthreads();

    uchar val = 0;
    if (y < img.rows && x < img.cols) {
        const unsigned int img_index = y * img.step + x;
        val = (!binary || img.data[img_index] == 0) ? img.data[img_index] : 1;
    }

    //const unsigned long x2 = x * x, x3 = x2 * x;
    //const unsigned long y2 = y * y, y3 = y2 * y;
    T res = warpButterflyReduce(static_cast<T>(val));
    if (res) {
        const T x1 = x - cX, x2 = x1 * x1, x3 = x2 * x1; // might be quicker to do x1*x1*x1, let the compiler decide?
        const T y1 = y - cY, y2 = y1 * y1, y3 = y2 * y1;
        smem[threadIdx.y][0] = warpButterflyReduce(x2 * static_cast<T>(val));
        smem[threadIdx.y][1] = warpButterflyReduce(x1 * y1 * static_cast<T>(val));
        smem[threadIdx.y][2] = y2 * res;
        smem[threadIdx.y][3] = warpButterflyReduce(x3 * static_cast<T>(val));
        smem[threadIdx.y][4] = warpButterflyReduce(x2 * y1 * static_cast<T>(val));
        smem[threadIdx.y][5] = warpButterflyReduce(x1 * y2 * static_cast<T>(val));
        smem[threadIdx.y][6] = y3 * res;
    }
    __syncthreads();

    // blockSizeY - this has to be 16 as below we are using half warp reduce and above we have to enforce 32 for warp reduce
    if (threadIdx.x < blockSizeY && threadIdx.y < n)
        smem[threadIdx.y][0] = halfWarpButterflyReduce(smem[threadIdx.x][threadIdx.y]);
    __syncthreads();

    if (threadIdx.y == 0 && threadIdx.x < n) {
        const T tmp = smem[threadIdx.x][0];
        if (tmp)
            atomicAdd(&moments[threadIdx.x], tmp);
    }
    //atomicAdd(&moments[threadIdx.x], smem[threadIdx.x][0]);
}




template <typename T>
__global__ void ComputeCentralMomentsShared(const cuda::PtrStepSzb img, bool binary, const T* m00, const T* m10, const T* m01, T* moments) {


    //if (*m00 == 0 || *m10 == 0 || *m01 == 0)
    //    return;

    const T cX = *m10 / *m00;
    const T cY = *m01 / *m00;
    if (!cX && !cY) return;

    const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    //const T x1 = x - cX, y1 = y - cY;
    constexpr int n = 7;
    __shared__ T smem[16][n];

    if (threadIdx.y < n && threadIdx.x < 16)
        smem[threadIdx.x][threadIdx.y] = 0;
    __syncthreads();

    uchar val = 0;
    if (y < img.rows && x < img.cols) {
        const unsigned int img_index = y * img.step + x;
        val = (!binary || img.data[img_index] == 0) ? img.data[img_index] : 1;
    }

    //const unsigned long x2 = x * x, x3 = x2 * x;
    //const unsigned long y2 = y * y, y3 = y2 * y;
    T res = warpButterflyReduce(static_cast<T>(val));
    if (res) {
        const T x1 = x - cX, x2 = x1 * x1, x3 = x2 * x1;
        const T y1 = y - cY, y2 = y1 * y1, y3 = y2 * y1;
        smem[threadIdx.y][0] = warpButterflyReduce(x2 * static_cast<T>(val));
        smem[threadIdx.y][1] = warpButterflyReduce(x1 * y1 * static_cast<T>(val));
        smem[threadIdx.y][2] = y2 * res;
        smem[threadIdx.y][3] = warpButterflyReduce(x3 * static_cast<T>(val));
        smem[threadIdx.y][4] = warpButterflyReduce(x2 * y1 * static_cast<T>(val));
        smem[threadIdx.y][5] = warpButterflyReduce(x1 * y2 * static_cast<T>(val));
        smem[threadIdx.y][6] = y3 * res;
    }
    __syncthreads();

    // blockSizeY - this has to be 16 as below we are using half warp reduce and above we have to enforce 32 for warp reduce
    if (threadIdx.x < blockSizeY && threadIdx.y < n)
        smem[threadIdx.y][0] = halfWarpButterflyReduce(smem[threadIdx.x][threadIdx.y]);
    __syncthreads();

    if (threadIdx.y == 0 && threadIdx.x < n) {
        const T tmp = smem[threadIdx.x][0];
        if(tmp)
            atomicAdd(&moments[threadIdx.x], tmp);
    }
        //atomicAdd(&moments[threadIdx.x], smem[threadIdx.x][0]);
}


template<typename T>
__global__ void ComputeCenteroid1(T* moments) {
    moments[17] = moments[1] / moments[0];
    moments[18] = moments[2] / moments[0];
}



__global__ void ComputeCenteroid(const double* moments, double2* centroid) {
    centroid->x = moments[m10] / moments[m00];
    centroid->y = moments[m01] / moments[m00];
}

__global__ void ComputeCenteralMoments(const cuda::PtrStepSzb img, bool binary,
                                       const double2* centroid, double* moments) {
    const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (y < img.rows && x < img.cols) {
      const unsigned int img_index = y * img.step + x;
      const unsigned char val = (!binary || img.data[img_index] == 0) ? img.data[img_index] : 1;
      if (val > 0) {
        const double x1 = x - centroid->x, x2 = x1 * x1, x3 = x2 * x1;
        const double y1 = y - centroid->y, y2 = y1 * y1, y3 = y2 * y1;

        atomicAdd(&moments[mu20], x2      * val);
        atomicAdd(&moments[mu11], x1 * y1 * val);
        atomicAdd(&moments[mu02],      y2 * val);
        atomicAdd(&moments[mu30], x3      * val);
        atomicAdd(&moments[mu21], x2 * y1 * val);
        atomicAdd(&moments[mu12], x1 * y2 * val);
        atomicAdd(&moments[mu03],      y3 * val);
      }
    }
}

void ComputeCenteralNormalizedMoments(cv::Moments& moments_cpu) {
    const double m00_pow2 = pow(moments_cpu.m00, 2), m00_pow2p5 = pow(moments_cpu.m00, 2.5);

    moments_cpu.nu20 = moments_cpu.mu20 / m00_pow2;
    moments_cpu.nu11 = moments_cpu.mu11 / m00_pow2;
    moments_cpu.nu02 = moments_cpu.mu02 / m00_pow2;
    moments_cpu.nu30 = moments_cpu.mu30 / m00_pow2p5;
    moments_cpu.nu21 = moments_cpu.mu21 / m00_pow2p5;
    moments_cpu.nu12 = moments_cpu.mu12 / m00_pow2p5;
    moments_cpu.nu03 = moments_cpu.mu03 / m00_pow2p5;
}

void Benchmark(const int idx, const cv::cuda::GpuMat& img, bool binary) {
    dim3 blockSize(blockSizeX, blockSizeY, 1);
    dim3 gridSize((img.cols + blockSize.x - 1) / blockSize.x, (img.rows + blockSize.y - 1) / blockSize.y, 1);
    cuda::Stream stream;
    cuda::Event start, end;

    // calculate gs result
    GpuMat momentsGpuGs = GpuMat(1, momentsSize, CV_64F, cv::Scalar(0));
    double2* centroid;
    cudaSafeCall(hipMalloc(&centroid, sizeof(double2)));
    start.record(stream);
    ComputeSpatialMoments << <gridSize, blockSize, 0, cuda::StreamAccessor::getStream(stream) >> > (img, binary, momentsGpuGs.ptr<double>(0));
    ComputeCenteroid << <dim3(1, 1, 1), dim3(1, 1, 1), 0, cuda::StreamAccessor::getStream(stream) >> > (momentsGpuGs.ptr<double>(0), centroid);
    ComputeCenteralMoments << <gridSize, blockSize, 0, cuda::StreamAccessor::getStream(stream) >> > (img, binary, centroid, momentsGpuGs.ptr<double>(0));
    end.record(stream);
    stream.waitForCompletion();
    const float nsGs = Event::elapsedTime(start, end) * 1000;
    Mat momentsCpuGs; momentsGpuGs.download(momentsCpuGs);
    cudaSafeCall(hipFree(centroid));

    GpuMat momentsGpu, momentsGpu64F;
    switch (idx) {
        case 0:
        {
            printf("\nOriginal using double\n");
            momentsGpu = GpuMat(1, momentsSize, CV_64F, cv::Scalar(0));
            start.record(stream);
            ComputeSpatialMoments << <gridSize, blockSize, 0, cuda::StreamAccessor::getStream(stream) >> > (img, binary, momentsGpu.ptr<double>(0));
            end.record(stream);
            momentsGpu64F = momentsGpu;
            break;
        }
        case 1:
        {
            printf("\nOriginal using float\n");
            momentsGpu = GpuMat(1, momentsSize, CV_32F, cv::Scalar(0));
            start.record(stream);
            ComputeSpatialMoments << <gridSize, blockSize, 0, cuda::StreamAccessor::getStream(stream) >> > (img, binary, momentsGpu.ptr<float>(0));
            end.record(stream);
            momentsGpu.convertTo(momentsGpu64F, CV_64F);
            break;
        }
        case 2:
        {
            printf("\nShared memory with partial reduction using double\n");
            momentsGpu = GpuMat(1, momentsSize, CV_64F, cv::Scalar(0));
            start.record(stream);
            ComputeSpatialMomentsSharedPartialReduction << <gridSize, blockSize, 0, cuda::StreamAccessor::getStream(stream) >> > (img, binary, momentsGpu.ptr<double>(0));
            end.record(stream);
            momentsGpu64F = momentsGpu;
            break;
        }
        case 3:
        {
            printf("\nShared memory with partial reduction using float\n");
            momentsGpu = GpuMat(1, momentsSize, CV_32F, cv::Scalar(0));
            start.record(stream);
            ComputeSpatialMomentsSharedPartialReduction << <gridSize, blockSize, 0, cuda::StreamAccessor::getStream(stream) >> > (img, binary, momentsGpu.ptr<float>(0));
            end.record(stream);
            momentsGpu.convertTo(momentsGpu64F, CV_64F);
            break;
        }
        case 4:
        {
            printf("\nShared memory with full reduction using double\n");
            momentsGpu = GpuMat(1, momentsSize, CV_64F, cv::Scalar(0));
            start.record(stream);
            ComputeSpatialMomentsSharedFullReduction << <gridSize, blockSize, 0, cuda::StreamAccessor::getStream(stream) >> > (img, binary, momentsGpu.ptr<double>(0));
            end.record(stream);
            momentsGpu64F = momentsGpu;
            break;
        }
        case 5:
        {
            printf("\nShared memory with full reduction using float\n");
            momentsGpu = GpuMat(1, momentsSize, CV_32F, cv::Scalar(0));
            start.record(stream);
            ComputeSpatialMomentsSharedFullReduction << <gridSize, blockSize, 0, cuda::StreamAccessor::getStream(stream) >> > (img, binary, momentsGpu.ptr<float>(0));
            end.record(stream);
            momentsGpu.convertTo(momentsGpu64F, CV_64F);
            break;
        }
        case 6:
        {
            printf("\nShared memory with full reduction using float S1\n");
            momentsGpu = GpuMat(1, momentsSize, CV_32F, cv::Scalar(0));
            start.record(stream);
            ComputeSpatialMomentsSharedFullReductionS1 << <gridSize, blockSize, 0, cuda::StreamAccessor::getStream(stream) >> > (img, binary, momentsGpu.ptr<float>(0));
            end.record(stream);
            momentsGpu.convertTo(momentsGpu64F, CV_64F);
            break;
        }
        case 7:
        {
            printf("\nCentral Moments Shared memory with full reduction using double\n");
            momentsGpu = GpuMat(1, momentsSize, CV_64F, cv::Scalar(0));
            start.record(stream);
            ComputeSpatialMomentsSharedFullReductionS1 << <gridSize, blockSize, 0, cuda::StreamAccessor::getStream(stream) >> > (img, binary, momentsGpu.ptr<double>(0));

            // should we pass pointer or ptrstepsz?
            ComputeCenteroid1 << < dim3(1, 1, 1), dim3(1, 1, 1), 0, cuda::StreamAccessor::getStream(stream) >> > (momentsGpu.ptr<double>(0));
            //ComputeCentralMomentsShared << <gridSize, blockSize, 0, cuda::StreamAccessor::getStream(stream) >> > (img, binary, momentsGpu.ptr<double>(0), momentsGpu.ptr<double>(0)+1, momentsGpu.ptr<double>(0)+2, momentsGpu.ptr<double>(0)+10);
            ComputeCentralMomentsShared1 << <gridSize, blockSize, 0, cuda::StreamAccessor::getStream(stream) >> > (img, binary, momentsGpu.ptr<double>(0) + 17, momentsGpu.ptr<double>(0) + 10);
            end.record(stream);
            //momentsGpu.convertTo(momentsGpu64F, CV_64F);
            momentsGpu64F = momentsGpu;
            //momentsGpu.convertTo(momentsGpu64F, CV_64F);
            break;
        }
        case 8:
        {
            printf("\nCentral Moments Shared memory with full reduction using float\n");
            momentsGpu = GpuMat(1, momentsSize, CV_32F, cv::Scalar(0));
            start.record(stream);
            ComputeSpatialMomentsSharedFullReductionS1 << <gridSize, blockSize, 0, cuda::StreamAccessor::getStream(stream) >> > (img, binary, momentsGpu.ptr<float>(0));
            ComputeCentralMomentsShared << <gridSize, blockSize, 0, cuda::StreamAccessor::getStream(stream) >> > (img, binary, momentsGpu.ptr<float>(0), momentsGpu.ptr<float>(0) + 1, momentsGpu.ptr<float>(0) + 2, momentsGpu.ptr<float>(0) + 10);
            end.record(stream);
            momentsGpu.convertTo(momentsGpu64F, CV_64F);
            break;
        }
        case 9:
        {
            printf("\nShared memory with full reduction using double and coalecsed reads\n");
            blockSize = dim3(blockSizeX, blockSizeY, 1);
            gridSize = dim3(divUp(img.cols/4, blockSizeX), divUp(img.rows, blockSizeY));
            momentsGpu = GpuMat(1, momentsSize, CV_64F, cv::Scalar(0));
            start.record(stream);
            ComputeSpatialMomentsSharedFullReductionCoaleced << <gridSize, blockSize, 0, cuda::StreamAccessor::getStream(stream) >> > (img, binary, momentsGpu.ptr<double>(0));
            end.record(stream);
            momentsGpu64F = momentsGpu;
            break;
        }
        case 10:
        {
            printf("\nShared memory with full reduction using float and coalecsed reads\n");
            blockSize = dim3(blockSizeX, blockSizeY, 1);
            gridSize = dim3(divUp(img.cols / 4, blockSizeX), divUp(img.rows, blockSizeY));
            momentsGpu = GpuMat(1, momentsSize, CV_32F, cv::Scalar(0));
            start.record(stream);
            ComputeSpatialMomentsSharedFullReductionCoaleced << <gridSize, blockSize, 0, cuda::StreamAccessor::getStream(stream) >> > (img, binary, momentsGpu.ptr<float>(0));
            end.record(stream);
            momentsGpu.convertTo(momentsGpu64F, CV_64F);
            break;
        }
        case 11:
        {
            printf("\nCentral Moments memory with full coalesced reduction using double\n");
            blockSize = dim3(blockSizeX, blockSizeY, 1);
            gridSize = dim3(divUp(img.cols / 4, blockSizeX), divUp(img.rows, blockSizeY));
            momentsGpu = GpuMat(1, momentsSize, CV_64F, cv::Scalar(0));
            start.record(stream);
            ComputeSpatialMomentsSharedFullReductionCoaleced << <gridSize, blockSize, 0, cuda::StreamAccessor::getStream(stream) >> > (img, binary, momentsGpu.ptr<double>(0));
            ComputeCentralMomentsSharedUchar << <gridSize, blockSize, 0, cuda::StreamAccessor::getStream(stream) >> > (img, binary, momentsGpu.ptr<double>(0), momentsGpu.ptr<double>(0) + 1, momentsGpu.ptr<double>(0) + 2, momentsGpu.ptr<double>(0) + 10);
            end.record(stream);
            momentsGpu64F = momentsGpu;
            break;
        }
        case 12:
        {
            printf("\nCentral Moments Shared memory with full coalesced reduction using float\n");
            blockSize = dim3(blockSizeX, blockSizeY, 1);
            gridSize = dim3(divUp(img.cols / 4, blockSizeX), divUp(img.rows, blockSizeY));
            momentsGpu = GpuMat(1, momentsSize, CV_32F, cv::Scalar(0));
            start.record(stream);
            //ComputeSpatialMomentsSharedFullReductionS1 << <gridSize, blockSize, 0, cuda::StreamAccessor::getStream(stream) >> > (img, binary, momentsGpu.ptr<float>(0));

            ComputeSpatialMomentsSharedFullReductionCoaleced << <gridSize, blockSize, 0, cuda::StreamAccessor::getStream(stream) >> > (img, binary, momentsGpu.ptr<float>(0));
            ComputeCentralMomentsSharedUchar << <gridSize, blockSize, 0, cuda::StreamAccessor::getStream(stream) >> > (img, binary, momentsGpu.ptr<float>(0), momentsGpu.ptr<float>(0) + 1, momentsGpu.ptr<float>(0) + 2, momentsGpu.ptr<float>(0) + 10);
            end.record(stream);
            momentsGpu.convertTo(momentsGpu64F, CV_64F);
            break;
        }
    }

    stream.waitForCompletion();
    const float ns = Event::elapsedTime(start, end)*1000;
    printf("  eltime - %.2fus (GS: %.2fus), speedup %.2fX\n", ns, nsGs, nsGs/ns);

    Mat momentsCpu; momentsGpu64F.download(momentsCpu);
    double cumErr = 0;
    for (int i = 0; i < 17; i++) {
        printf("%f, %f\n", momentsCpuGs.at<double>(i), momentsCpu.at<double>(i));
        cumErr += abs(momentsCpuGs.at<double>(i) - momentsCpu.at<double>(i));
    }
    if (cumErr != 0)
        printf("  cumulative error %f\n", cumErr);

}

//enum MomentType {
//    SPATIAL,
//    CENTRAL
//};

template <typename T>
void Moments1(const PtrStepSzb src, PtrStep<T> moments, bool binary, bool computeCentral, hipStream_t stream) {
    dim3 blockSize = dim3(blockSizeX, blockSizeY);
    dim3 gridSize = dim3(divUp(src.cols, blockSizeX), divUp(src.rows, blockSizeY));
    ComputeSpatialMomentsSharedFullReductionS1 << <gridSize, blockSize, 0, stream >> > (src, binary, moments.ptr());
    if (computeCentral) {
        //ComputeCentralMomentsShared << <gridSize, blockSize, 0, stream >> > (src, binary, moments.ptr(), moments.ptr() + 1, moments.ptr() + 2, moments.ptr() + 10);
        ComputeCenteroid1 << < dim3(1, 1, 1), dim3(1, 1, 1), 0, stream >> > (moments.ptr());
        //ComputeCentralMomentsShared << <gridSize, blockSize, 0, cuda::StreamAccessor::getStream(stream) >> > (img, binary, momentsGpu.ptr<double>(0), momentsGpu.ptr<double>(0)+1, momentsGpu.ptr<double>(0)+2, momentsGpu.ptr<double>(0)+10);
        ComputeCentralMomentsShared1 << <gridSize, blockSize, 0, stream >> > (src, binary, moments.ptr()+17, moments.ptr()+10);
    }

    if (stream == 0)
        cudaSafeCall(hipDeviceSynchronize());

    // moments can be float or double
    // can we request calculation to be float for spatial and double for the other
    // need to request type of calc

    // if spatial just do one
    // central both

    // all - need to normalize forget
    // need a helper routine which downloads and normalizes the result


}

template void Moments1<float>(const PtrStepSzb src, PtrStep<float> moments, bool binary, bool computeCentral, hipStream_t stream);
template void Moments1<double>(const PtrStepSzb src, PtrStep<double> moments, bool binary, bool computeCentral, hipStream_t stream);

cv::Moments Moments(const cv::cuda::GpuMat& img, bool binary) {

    const dim3 blockSize(blockSizeX, blockSizeY, 1);
    const dim3 gridSize((img.cols + blockSize.x - 1) / blockSize.x,
                        (img.rows + blockSize.y - 1) / blockSize.y, 1);

    double2* centroid;
    cudaSafeCall(hipMalloc(&centroid, sizeof(double2)));
    cv::cuda::GpuMat moments_gpu(1, momentsSize, CV_64F, cv::Scalar(0));
    ComputeSpatialMoments <<<gridSize, blockSize>>>(img, binary, moments_gpu.ptr<double>(0));
    cudaSafeCall(hipGetLastError());

    ComputeCenteroid <<<dim3(1, 1, 1), dim3(1, 1, 1)>>>(moments_gpu.ptr<double>(0), centroid);
    cudaSafeCall(hipGetLastError());

    ComputeCenteralMoments <<<gridSize, blockSize>>>(img, binary, centroid, moments_gpu.ptr<double>(0));
    cudaSafeCall(hipFree(centroid));
    cudaSafeCall(hipGetLastError());

    cv::Moments moments_cpu;
    cv::Mat moments_map(1, momentsSize, CV_64F, reinterpret_cast<double*>(&moments_cpu));
    moments_gpu.download(moments_map);
    cudaSafeCall(hipDeviceSynchronize());

    ComputeCenteralNormalizedMoments(moments_cpu);

    //for (int i = 0; i < 11; i++)
    //    Benchmark(i, img, binary);

    //Benchmark(8, img, binary);
    ////Benchmark(12, img, binary);
    //Benchmark(7, img, binary);
    //Benchmark(4, img, binary);
    return moments_cpu;
}

}}}}


#endif /* CUDA_DISABLER */
