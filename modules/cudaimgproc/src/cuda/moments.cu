#include "hip/hip_runtime.h"
// This file is part of OpenCV project.
// It is subject to the license terms in the LICENSE file found in the top-level directory
// of this distribution and at http://opencv.org/license.html.

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/emulation.hpp"
#include "opencv2/core/cuda/transform.hpp"
#include "opencv2/core/cuda/functional.hpp"
#include "opencv2/core/cuda/utility.hpp"
#include "opencv2/core/cuda.hpp"

using namespace cv::cuda;
using namespace cv::cuda::device;


namespace cv { namespace cuda { namespace device { namespace imgproc {

constexpr int blockSizeX = 16;
constexpr int blockSizeY = 16;
constexpr int momentsSize = sizeof(cv::Moments) / sizeof(double);

constexpr int m00 = offsetof(cv::Moments, m00) / sizeof(double);
constexpr int m10 = offsetof(cv::Moments, m10) / sizeof(double);
constexpr int m01 = offsetof(cv::Moments, m01) / sizeof(double);
constexpr int m20 = offsetof(cv::Moments, m20) / sizeof(double);
constexpr int m11 = offsetof(cv::Moments, m11) / sizeof(double);
constexpr int m02 = offsetof(cv::Moments, m02) / sizeof(double);
constexpr int m30 = offsetof(cv::Moments, m30) / sizeof(double);
constexpr int m21 = offsetof(cv::Moments, m21) / sizeof(double);
constexpr int m12 = offsetof(cv::Moments, m12) / sizeof(double);
constexpr int m03 = offsetof(cv::Moments, m03) / sizeof(double);

constexpr int mu20 = offsetof(cv::Moments, mu20) / sizeof(double);
constexpr int mu11 = offsetof(cv::Moments, mu11) / sizeof(double);
constexpr int mu02 = offsetof(cv::Moments, mu02) / sizeof(double);
constexpr int mu30 = offsetof(cv::Moments, mu30) / sizeof(double);
constexpr int mu21 = offsetof(cv::Moments, mu21) / sizeof(double);
constexpr int mu12 = offsetof(cv::Moments, mu12) / sizeof(double);
constexpr int mu03 = offsetof(cv::Moments, mu03) / sizeof(double);

__global__ void ComputeSpatialMoments(const cuda::PtrStepSzb img, bool binary, double* moments) {
    const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (y < img.rows && x < img.cols) {
      const unsigned int img_index = y * img.step + x;
      const unsigned char val = (!binary || img.data[img_index] == 0) ? img.data[img_index] : 1;
      if (val > 0) {
        const unsigned long x2 = x * x, x3 = x2 * x;
        const unsigned long y2 = y * y, y3 = y2 * y;

        atomicAdd(&moments[m00],           val);
        atomicAdd(&moments[m10], x       * val);
        atomicAdd(&moments[m01],      y  * val);
        atomicAdd(&moments[m20], x2      * val);
        atomicAdd(&moments[m11], x  * y  * val);
        atomicAdd(&moments[m02],      y2 * val);
        atomicAdd(&moments[m30], x3      * val);
        atomicAdd(&moments[m21], x2 * y  * val);
        atomicAdd(&moments[m12], x  * y2 * val);
        atomicAdd(&moments[m03],      y3 * val);
      }
    }
}

__global__ void ComputeCenteroid(const double* moments, double2* centroid) {
    centroid->x = moments[m10] / moments[m00];
    centroid->y = moments[m01] / moments[m00];
}

__global__ void ComputeCenteralMoments(const cuda::PtrStepSzb img, bool binary,
                                       const double2* centroid, double* moments) {
    const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (y < img.rows && x < img.cols) {
      const unsigned int img_index = y * img.step + x;
      const unsigned char val = (!binary || img.data[img_index] == 0) ? img.data[img_index] : 1;
      if (val > 0) {
        const double x1 = x - centroid->x, x2 = x1 * x1, x3 = x2 * x1;
        const double y1 = y - centroid->y, y2 = y1 * y1, y3 = y2 * y1;

        atomicAdd(&moments[mu20], x2      * val);
        atomicAdd(&moments[mu11], x1 * y1 * val);
        atomicAdd(&moments[mu02],      y2 * val);
        atomicAdd(&moments[mu30], x3      * val);
        atomicAdd(&moments[mu21], x2 * y1 * val);
        atomicAdd(&moments[mu12], x1 * y2 * val);
        atomicAdd(&moments[mu03],      y3 * val);
      }
    }
}

void ComputeCenteralNormalizedMoments(cv::Moments& moments_cpu) {
    const double m00_pow2 = pow(moments_cpu.m00, 2), m00_pow2p5 = pow(moments_cpu.m00, 2.5);

    moments_cpu.nu20 = moments_cpu.mu20 / m00_pow2;
    moments_cpu.nu11 = moments_cpu.mu11 / m00_pow2;
    moments_cpu.nu02 = moments_cpu.mu02 / m00_pow2;
    moments_cpu.nu30 = moments_cpu.mu30 / m00_pow2p5;
    moments_cpu.nu21 = moments_cpu.mu21 / m00_pow2p5;
    moments_cpu.nu12 = moments_cpu.mu12 / m00_pow2p5;
    moments_cpu.nu03 = moments_cpu.mu03 / m00_pow2p5;
}

cv::Moments Moments(const cv::cuda::GpuMat& img, bool binary) {
    const dim3 blockSize(blockSizeX, blockSizeY, 1);
    const dim3 gridSize((img.cols + blockSize.x - 1) / blockSize.x,
                        (img.rows + blockSize.y - 1) / blockSize.y, 1);

    double2* centroid;
    cudaSafeCall(hipMalloc(&centroid, sizeof(double2)));
    cv::cuda::GpuMat moments_gpu(1, momentsSize, CV_64F, cv::Scalar(0));
    ComputeSpatialMoments <<<gridSize, blockSize>>>(img, binary, moments_gpu.ptr<double>(0));
    cudaSafeCall(hipGetLastError());

    ComputeCenteroid <<<dim3(1, 1, 1), dim3(1, 1, 1)>>>(moments_gpu.ptr<double>(0), centroid);
    cudaSafeCall(hipGetLastError());

    ComputeCenteralMoments <<<gridSize, blockSize>>>(img, binary, centroid, moments_gpu.ptr<double>(0));
    cudaSafeCall(hipFree(centroid));
    cudaSafeCall(hipGetLastError());

    cv::Moments moments_cpu;
    cv::Mat moments_map(1, momentsSize, CV_64F, reinterpret_cast<double*>(&moments_cpu));
    moments_gpu.download(moments_map);
    cudaSafeCall(hipDeviceSynchronize());

    ComputeCenteralNormalizedMoments(moments_cpu);

    return moments_cpu;
}

}}}}


#endif /* CUDA_DISABLER */
