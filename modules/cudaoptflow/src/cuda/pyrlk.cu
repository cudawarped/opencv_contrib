#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/utility.hpp"
#include "opencv2/core/cuda/functional.hpp"
#include "opencv2/core/cuda/limits.hpp"
#include "opencv2/core/cuda/vec_math.hpp"
#include "opencv2/core/cuda/reduce.hpp"
#include "opencv2/core/cuda/filters.hpp"
#include "opencv2/core/cuda/border_interpolate.hpp"
#include "opencv2/cudev.hpp"

#include <iostream>

using namespace cv::cuda;
using namespace cv::cuda::device;

namespace pyrlk
{
    __constant__ int c_winSize_x;
    __constant__ int c_winSize_y;
    __constant__ int c_halfWin_x;
    __constant__ int c_halfWin_y;
    __constant__ int c_iters;

    //texture<uchar, hipTextureType2D, hipReadModeNormalizedFloat> tex_I8U(false, hipFilterModeLinear, hipAddressModeClamp);
    //texture<uchar4, hipTextureType2D, hipReadModeNormalizedFloat> tex_I8UC4(false, hipFilterModeLinear, hipAddressModeClamp);

    //texture<ushort4, hipTextureType2D, hipReadModeNormalizedFloat> tex_I16UC4(false, hipFilterModeLinear, hipAddressModeClamp);


    //texture<float, hipTextureType2D, hipReadModeElementType> tex_If(false, hipFilterModeLinear, hipAddressModeClamp);
    //texture<float4, hipTextureType2D, hipReadModeElementType> tex_If4(false, hipFilterModeLinear, hipAddressModeClamp);

    ////texture<uchar, hipTextureType2D, hipReadModeElementType> tex_Ib(false, hipFilterModePoint, hipAddressModeClamp);

    //texture<uchar, hipTextureType2D, hipReadModeNormalizedFloat> tex_J8U(false, hipFilterModeLinear, hipAddressModeClamp);
    //texture<uchar4, hipTextureType2D, hipReadModeNormalizedFloat> tex_J8UC4(false, hipFilterModeLinear, hipAddressModeClamp);

    //texture<ushort4, hipTextureType2D, hipReadModeNormalizedFloat> tex_J16UC4(false, hipFilterModeLinear, hipAddressModeClamp);


    //texture<float, hipTextureType2D, hipReadModeElementType> tex_Jf(false, hipFilterModeLinear, hipAddressModeClamp);
    //texture<float4, hipTextureType2D, hipReadModeElementType> tex_Jf4(false, hipFilterModeLinear, hipAddressModeClamp);


    //template <int cn, typename T> struct Tex_I
    //{
    //    static __host__ __forceinline__ void bindTexture_(PtrStepSz<typename TypeVec<T, cn>::vec_type> I)
    //    {
    //        CV_UNUSED(I);
    //    }
    //};

    //template <> struct Tex_I<1, uchar>
    //{
    //    static __device__ __forceinline__ float read(float x, float y)
    //    {
    //        return tex2D(tex_I8U, x, y);
    //    }
    //    static __host__ __forceinline__ void bindTexture_(PtrStepSz<uchar>& I)
    //    {
    //        bindTexture(&tex_I8U, I);
    //    }
    //};
    //template <> struct Tex_I<1, ushort>
    //{
    //    static __device__ __forceinline__ float read(float x, float y)
    //    {
    //        return 0.0;
    //    }
    //    static __host__ __forceinline__ void bindTexture_(PtrStepSz<ushort>& I)
    //    {
    //        CV_UNUSED(I);
    //    }
    //};
    //template <> struct Tex_I<1, int>
    //{
    //    static __device__ __forceinline__ float read(float x, float y)
    //    {
    //        return 0.0;
    //    }
    //    static __host__ __forceinline__ void bindTexture_(PtrStepSz<int>& I)
    //    {
    //        CV_UNUSED(I);
    //    }
    //};
    //template <> struct Tex_I<1, float>
    //{
    //    static __device__ __forceinline__ float read(float x, float y)
    //    {
    //        return tex2D(tex_If, x, y);
    //    }
    //    static __host__ __forceinline__ void bindTexture_(PtrStepSz<float>& I)
    //    {
    //        bindTexture(&tex_If, I);
    //    }
    //};
    //// ****************** 3 channel specializations ************************
    //template <> struct Tex_I<3, uchar>
    //{
    //    static __device__ __forceinline__ float3 read(float x, float y)
    //    {
    //        return make_float3(0,0,0);
    //    }
    //    static __host__ __forceinline__ void bindTexture_(PtrStepSz<uchar3> I)
    //    {
    //        CV_UNUSED(I);
    //    }
    //};
    //template <> struct Tex_I<3, ushort>
    //{
    //    static __device__ __forceinline__ float3 read(float x, float y)
    //    {
    //        return make_float3(0, 0, 0);
    //    }
    //    static __host__ __forceinline__ void bindTexture_(PtrStepSz<ushort3> I)
    //    {
    //        CV_UNUSED(I);
    //    }
    //};
    //template <> struct Tex_I<3, int>
    //{
    //    static __device__ __forceinline__ float3 read(float x, float y)
    //    {
    //        return make_float3(0, 0, 0);
    //    }
    //    static __host__ __forceinline__ void bindTexture_(PtrStepSz<int3> I)
    //    {
    //        CV_UNUSED(I);
    //    }
    //};
    //template <> struct Tex_I<3, float>
    //{
    //    static __device__ __forceinline__ float3 read(float x, float y)
    //    {
    //        return make_float3(0, 0, 0);
    //    }
    //    static __host__ __forceinline__ void bindTexture_(PtrStepSz<float3> I)
    //    {
    //        CV_UNUSED(I);
    //    }
    //};
    //// ****************** 4 channel specializations ************************

    //template <> struct Tex_I<4, uchar>
    //{
    //    static __device__ __forceinline__ float4 read(float x, float y)
    //    {
    //        return tex2D(tex_I8UC4, x, y);
    //    }
    //    static __host__ __forceinline__ void bindTexture_(PtrStepSz<uchar4>& I)
    //    {
    //        bindTexture(&tex_I8UC4, I);
    //    }
    //};
    //template <> struct Tex_I<4, ushort>
    //{
    //    static __device__ __forceinline__ float4 read(float x, float y)
    //    {
    //        return tex2D(tex_I16UC4, x, y);
    //    }
    //    static __host__ __forceinline__ void bindTexture_(PtrStepSz<ushort4>& I)
    //    {
    //        bindTexture(&tex_I16UC4, I);
    //    }
    //};
    //template <> struct Tex_I<4, float>
    //{
    //    static __device__ __forceinline__ float4 read(float x, float y)
    //    {
    //        return tex2D(tex_If4, x, y);
    //    }
    //    static __host__ __forceinline__ void bindTexture_(PtrStepSz<float4>& I)
    //    {
    //        bindTexture(&tex_If4, I);
    //    }
    //};
    //// ************* J  ***************
    //template <int cn, typename T> struct Tex_J
    //{
    //    static __host__ __forceinline__ void bindTexture_(PtrStepSz<typename TypeVec<T,cn>::vec_type>& J)
    //    {
    //        CV_UNUSED(J);
    //    }
    //};
    //template <> struct Tex_J<1, uchar>
    //{
    //    static __device__ __forceinline__ float read(float x, float y)
    //    {
    //        return tex2D(tex_J8U, x, y);
    //    }
    //    static __host__ __forceinline__ void bindTexture_(PtrStepSz<uchar>& J)
    //    {
    //        bindTexture(&tex_J8U, J);
    //    }
    //};
    //template <> struct Tex_J<1, float>
    //{
    //    static __device__ __forceinline__ float read(float x, float y)
    //    {
    //        return tex2D(tex_Jf, x, y);
    //    }
    //    static __host__ __forceinline__ void bindTexture_(PtrStepSz<float>& J)
    //    {
    //        bindTexture(&tex_Jf, J);
    //    }
    //};
    //// ************* 4 channel specializations ***************
    //template <> struct Tex_J<4, uchar>
    //{
    //    static __device__ __forceinline__ float4 read(float x, float y)
    //    {
    //        return tex2D(tex_J8UC4, x, y);
    //    }
    //    static __host__ __forceinline__ void bindTexture_(PtrStepSz<uchar4>& J)
    //    {
    //        bindTexture(&tex_J8UC4, J);
    //    }
    //};
    //template <> struct Tex_J<4, ushort>
    //{
    //    static __device__ __forceinline__ float4 read(float x, float y)
    //    {
    //        return tex2D(tex_J16UC4, x, y);
    //    }
    //    static __host__ __forceinline__ void bindTexture_(PtrStepSz<ushort4>& J)
    //    {
    //        bindTexture(&tex_J16UC4, J);
    //    }
    //};
    //template <> struct Tex_J<4, float>
    //{
    //    static __device__ __forceinline__ float4 read(float x, float y)
    //    {
    //        return tex2D(tex_Jf4, x, y);
    //    }
    //    static __host__ __forceinline__ void bindTexture_(PtrStepSz<float4>& J)
    //    {
    //        bindTexture(&tex_Jf4, J);
    //    }
    //};

    __device__ __forceinline__ void accum(float& dst, const float& val)
    {
        dst += val;
    }
    __device__ __forceinline__ void accum(float& dst, const float2& val)
    {
        dst += val.x + val.y;
    }
    __device__ __forceinline__ void accum(float& dst, const float3& val)
    {
        dst += val.x + val.y + val.z;
    }
    __device__ __forceinline__ void accum(float& dst, const float4& val)
    {
        dst += val.x + val.y + val.z + val.w;
    }

    __device__ __forceinline__ float abs_(float a)
    {
        return ::fabsf(a);
    }
    __device__ __forceinline__ float4 abs_(const float4& a)
    {
        return abs(a);
    }
    __device__ __forceinline__ float2 abs_(const float2& a)
    {
        return abs(a);
    }
    __device__ __forceinline__ float3 abs_(const float3& a)
    {
        return abs(a);
    }


    template<typename T> __device__ __forceinline__ typename TypeVec<float, 1>::vec_type ToFloat(const typename TypeVec<T, 1>::vec_type& other)
    {
        return other;
    }
    template<typename T> __device__ __forceinline__  typename TypeVec<float, 2>::vec_type ToFloat(const typename TypeVec<T, 2>::vec_type& other)
    {
        typename TypeVec<float, 2>::vec_type ret;
        ret.x = other.x;
        ret.y = other.y;
        return ret;
    }
    template<typename T> __device__ __forceinline__  typename TypeVec<float, 3>::vec_type ToFloat(const typename TypeVec<T, 3>::vec_type& other)
    {
        typename TypeVec<float, 3>::vec_type ret;
        ret.x = other.x;
        ret.y = other.y;
        ret.z = other.z;
        return ret;
    }
    template<typename T> __device__ __forceinline__  typename TypeVec<float, 4>::vec_type ToFloat(const typename TypeVec<T, 4>::vec_type& other)
    {
        typename TypeVec<float, 4>::vec_type ret;
        ret.x = other.x;
        ret.y = other.y;
        ret.z = other.z;
        ret.w = other.w;
        return ret;
    }

    template <typename T>
    struct DenormalizationFactor
    {
        static __device__ __forceinline__ float factor()
        {
            return 1.0f;
        }
    };

    template <>
    struct DenormalizationFactor<uchar>
    {
        static __device__ __forceinline__ float factor()
        {
            return 255.0f;
        }
    };

    //template <int cn, int PATCH_X, int PATCH_Y, bool calcErr, typename T>
    //__global__ void sparseKernel(cv::cudev::Texture<typename TypeVec<T, cn>::vec_type, typename TypeVec<float, cn>::vec_type>  texI, cv::cudev::Texture<typename TypeVec<T, cn>::vec_type, typename TypeVec<float, cn>::vec_type> texJ,
    //    const float2* prevPts, float2* nextPts, uchar* status, float* err, const int level, const int rows, const int cols)
    template <int cn, int PATCH_X, int PATCH_Y, bool calcErr, typename T, class Ptr2D>
    __global__ void sparseKernel(const Ptr2D texI, const Ptr2D texJ, const float2* prevPts, float2* nextPts, uchar* status, float* err, const int level, const int rows, const int cols)
    //__global__ void sparseKernel(const float2* prevPts, float2* nextPts, uchar* status, float* err,
    //    const int level, const int rows, const int cols)
    {
    #if __CUDA_ARCH__ <= 110
        const int BLOCK_SIZE = 128;
    #else
        const int BLOCK_SIZE = 256;
    #endif

        __shared__ float smem1[BLOCK_SIZE];
        __shared__ float smem2[BLOCK_SIZE];
        __shared__ float smem3[BLOCK_SIZE];

        const unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;

        float2 prevPt = prevPts[blockIdx.x];
        prevPt.x *= (1.0f / (1 << level));
        prevPt.y *= (1.0f / (1 << level));

        if (prevPt.x < 0 || prevPt.x >= cols || prevPt.y < 0 || prevPt.y >= rows)
        {
            if (tid == 0 && level == 0)
                status[blockIdx.x] = 0;

            return;
        }

        prevPt.x -= c_halfWin_x;
        prevPt.y -= c_halfWin_y;

        // extract the patch from the first image, compute covariation matrix of derivatives

        float A11 = 0;
        float A12 = 0;
        float A22 = 0;

        typedef typename TypeVec<float, cn>::vec_type work_type;

        work_type I_patch   [PATCH_Y][PATCH_X];
        work_type dIdx_patch[PATCH_Y][PATCH_X];
        work_type dIdy_patch[PATCH_Y][PATCH_X];

        for (int yBase = threadIdx.y, i = 0; yBase < c_winSize_y; yBase += blockDim.y, ++i)
        {
            for (int xBase = threadIdx.x, j = 0; xBase < c_winSize_x; xBase += blockDim.x, ++j)
            {
                float x = prevPt.x + xBase + 0.5f;
                float y = prevPt.y + yBase + 0.5f;

                //I_patch[i][j] = Tex_I<cn, T>::read(x, y);
                //typename TypeVec<T, cn>::vec_type tmp = texI(x, y);
                I_patch[i][j] = texI(y, x);
                //work_type tmp = texI(x, y);

                //printf("%d, %d",(float) I_patch[i][j], (float)tmp);
                //float4 tmp;
                //float4 tmp1;
                //tmp = tmp1;

                //tmp = I_patch[i][j];

                //I_patch[i][j] = tmp;

                //uchar4 tmp = texI(x, y);
                //float4 tmp1;
                //I_patch[i][j] = tmp1;
                //I_patch[i][j] = texI(x, y);

                // Scharr Deriv

                work_type dIdx = 3.0f * texI(y - 1, x + 1) + 10.0f * texI(y, x + 1) + 3.0f * texI(y + 1, x + 1) -
                    (3.0f * texI(y - 1, x - 1) + 10.0f * texI(y, x - 1) + 3.0f * texI(y + 1, x - 1));

                work_type dIdy = 3.0f * texI(y + 1, x - 1) + 10.0f * texI(y + 1, x) + 3.0f * texI(y + 1, x + 1) -
                    (3.0f * texI(y - 1, x - 1) + 10.0f * texI(y - 1, x) + 3.0f * texI(y - 1, x + 1));

                //work_type dIdx = 3.0f * texI(x + 1, y - 1) + 10.0f * texI(x + 1, y) + 3.0f * texI(x + 1, y + 1) -
                //    (3.0f * texI(x - 1, y - 1) + 10.0f * texI(x - 1, y) + 3.0f * texI(x - 1, y + 1));

                //work_type dIdy = 3.0f * texI(x - 1, y + 1) + 10.0f * texI(x, y + 1) + 3.0f * texI(x + 1, y + 1) -
                //    (3.0f * texI(x - 1, y - 1) + 10.0f * texI(x, y - 1) + 3.0f * texI(x + 1, y - 1));

                //work_type dIdx = 3.0f * Tex_I<cn,T>::read(x+1, y-1) + 10.0f * Tex_I<cn, T>::read(x+1, y) + 3.0f * Tex_I<cn,T>::read(x+1, y+1) -
                //                 (3.0f * Tex_I<cn,T>::read(x-1, y-1) + 10.0f * Tex_I<cn, T>::read(x-1, y) + 3.0f * Tex_I<cn,T>::read(x-1, y+1));

                //work_type dIdy = 3.0f * Tex_I<cn,T>::read(x-1, y+1) + 10.0f * Tex_I<cn, T>::read(x, y+1) + 3.0f * Tex_I<cn,T>::read(x+1, y+1) -
                //                (3.0f * Tex_I<cn,T>::read(x-1, y-1) + 10.0f * Tex_I<cn, T>::read(x, y-1) + 3.0f * Tex_I<cn,T>::read(x+1, y-1));

                dIdx_patch[i][j] = dIdx;
                dIdy_patch[i][j] = dIdy;

                accum(A11, dIdx * dIdx);
                accum(A12, dIdx * dIdy);
                accum(A22, dIdy * dIdy);
            }
        }

        reduce<BLOCK_SIZE>(smem_tuple(smem1, smem2, smem3), thrust::tie(A11, A12, A22), tid, thrust::make_tuple(plus<float>(), plus<float>(), plus<float>()));

    #if __CUDA_ARCH__ >= 300
        if (tid == 0)
        {
            smem1[0] = A11;
            smem2[0] = A12;
            smem3[0] = A22;
        }
    #endif

        __syncthreads();

        A11 = smem1[0];
        A12 = smem2[0];
        A22 = smem3[0];

        float D = A11 * A22 - A12 * A12;

        if (D < numeric_limits<float>::epsilon())
        {
            if (tid == 0 && level == 0)
                status[blockIdx.x] = 0;

            return;
        }

        D = 1.f / D;

        A11 *= D;
        A12 *= D;
        A22 *= D;

        float2 nextPt = nextPts[blockIdx.x];
        nextPt.x *= 2.f;
        nextPt.y *= 2.f;

        nextPt.x -= c_halfWin_x;
        nextPt.y -= c_halfWin_y;

        for (int k = 0; k < c_iters; ++k)
        {
            if (nextPt.x < -c_halfWin_x || nextPt.x >= cols || nextPt.y < -c_halfWin_y || nextPt.y >= rows)
            {
                if (tid == 0 && level == 0)
                    status[blockIdx.x] = 0;

                return;
            }

            float b1 = 0;
            float b2 = 0;

            for (int y = threadIdx.y, i = 0; y < c_winSize_y; y += blockDim.y, ++i)
            {
                for (int x = threadIdx.x, j = 0; x < c_winSize_x; x += blockDim.x, ++j)
                {
                    work_type I_val = I_patch[i][j];
                    //work_type J_val = Tex_J<cn, T>::read(nextPt.x + x + 0.5f, nextPt.y + y + 0.5f);
                    //work_type J_val = texJ(nextPt.x + x + 0.5f, nextPt.y + y + 0.5f);
                    work_type J_val = texJ(nextPt.y + y + 0.5f, nextPt.x + x + 0.5f);

                    work_type diff = (J_val - I_val) * 32.0f;

                    accum(b1, diff * dIdx_patch[i][j]);
                    accum(b2, diff * dIdy_patch[i][j]);
                }
            }

            reduce<BLOCK_SIZE>(smem_tuple(smem1, smem2), thrust::tie(b1, b2), tid, thrust::make_tuple(plus<float>(), plus<float>()));

        #if __CUDA_ARCH__ >= 300
            if (tid == 0)
            {
                smem1[0] = b1;
                smem2[0] = b2;
            }
        #endif

            __syncthreads();

            b1 = smem1[0];
            b2 = smem2[0];

            float2 delta;
            delta.x = A12 * b2 - A22 * b1;
            delta.y = A12 * b1 - A11 * b2;

            nextPt.x += delta.x;
            nextPt.y += delta.y;

            if (::fabs(delta.x) < 0.01f && ::fabs(delta.y) < 0.01f)
                break;
        }

        float errval = 0;
        if (calcErr)
        {
            for (int y = threadIdx.y, i = 0; y < c_winSize_y; y += blockDim.y, ++i)
            {
                for (int x = threadIdx.x, j = 0; x < c_winSize_x; x += blockDim.x, ++j)
                {
                    work_type I_val = I_patch[i][j];
                    //work_type J_val = Tex_J<cn, T>::read(nextPt.x + x + 0.5f, nextPt.y + y + 0.5f);
                    //work_type J_val = texJ(nextPt.x + x + 0.5f, nextPt.y + y + 0.5f);
                    work_type J_val = texJ(nextPt.y + y + 0.5f, nextPt.x + x + 0.5f);

                    work_type diff = J_val - I_val;

                    accum(errval, abs_(diff));
                }
            }

            reduce<BLOCK_SIZE>(smem1, errval, tid, plus<float>());
        }

        if (tid == 0)
        {
            nextPt.x += c_halfWin_x;
            nextPt.y += c_halfWin_y;

            nextPts[blockIdx.x] = nextPt;

            if (calcErr)
                err[blockIdx.x] = static_cast<float>(errval) / (::min(cn, 3) * c_winSize_x * c_winSize_y) * DenormalizationFactor<T>::factor();
        }
    }

    // Kernel, uses non texture fetches
    template <int PATCH_X, int PATCH_Y, bool calcErr, int cn, typename T, typename Ptr2D>
    __global__ void sparseKernel_(Ptr2D I, Ptr2D J, const float2* prevPts, float2* nextPts, uchar* status, float* err, const int level, const int rows, const int cols)
    {
#if __CUDA_ARCH__ <= 110
        const int BLOCK_SIZE = 128;
#else
        const int BLOCK_SIZE = 256;
#endif

        __shared__ float smem1[BLOCK_SIZE];
        __shared__ float smem2[BLOCK_SIZE];
        __shared__ float smem3[BLOCK_SIZE];

        const unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;

        float2 prevPt = prevPts[blockIdx.x];
        prevPt.x *= (1.0f / (1 << level));
        prevPt.y *= (1.0f / (1 << level));

        if (prevPt.x < 0 || prevPt.x >= cols || prevPt.y < 0 || prevPt.y >= rows)
        {
            if (tid == 0 && level == 0)
                status[blockIdx.x] = 0;

            return;
        }

        prevPt.x -= c_halfWin_x;
        prevPt.y -= c_halfWin_y;

        // extract the patch from the first image, compute covariation matrix of derivatives

        float A11 = 0;
        float A12 = 0;
        float A22 = 0;

        typedef typename TypeVec<float, cn>::vec_type work_type;

        work_type I_patch[PATCH_Y][PATCH_X];
        work_type dIdx_patch[PATCH_Y][PATCH_X];
        work_type dIdy_patch[PATCH_Y][PATCH_X];

        for (int yBase = threadIdx.y, i = 0; yBase < c_winSize_y; yBase += blockDim.y, ++i)
        {
            for (int xBase = threadIdx.x, j = 0; xBase < c_winSize_x; xBase += blockDim.x, ++j)
            {
                float x = prevPt.x + xBase + 0.5f;
                float y = prevPt.y + yBase + 0.5f;

                I_patch[i][j] = ToFloat<T>(I(y, x));

                // Scharr Deriv

                work_type dIdx = 3.0f * I(y - 1, x + 1) + 10.0f * I(y, x + 1) + 3.0f * I(y + 1, x + 1) -
                    (3.0f * I(y - 1, x - 1) + 10.0f * I(y, x - 1) + 3.0f * I(y + 1 , x - 1));

                work_type dIdy = 3.0f * I(y + 1, x - 1) + 10.0f * I(y + 1, x) + 3.0f * I(y+1, x + 1) -
                    (3.0f * I(y - 1, x - 1) + 10.0f * I(y-1, x) + 3.0f * I(y - 1, x + 1));

                dIdx_patch[i][j] = dIdx;
                dIdy_patch[i][j] = dIdy;

                accum(A11, dIdx * dIdx);
                accum(A12, dIdx * dIdy);
                accum(A22, dIdy * dIdy);
            }
        }

        reduce<BLOCK_SIZE>(smem_tuple(smem1, smem2, smem3), thrust::tie(A11, A12, A22), tid, thrust::make_tuple(plus<float>(), plus<float>(), plus<float>()));

#if __CUDA_ARCH__ >= 300
        if (tid == 0)
        {
            smem1[0] = A11;
            smem2[0] = A12;
            smem3[0] = A22;
        }
#endif

        __syncthreads();

        A11 = smem1[0];
        A12 = smem2[0];
        A22 = smem3[0];

        float D = A11 * A22 - A12 * A12;

        if (D < numeric_limits<float>::epsilon())
        {
            if (tid == 0 && level == 0)
                status[blockIdx.x] = 0;

            return;
        }

        D = 1.f / D;

        A11 *= D;
        A12 *= D;
        A22 *= D;

        float2 nextPt = nextPts[blockIdx.x];
        nextPt.x *= 2.f;
        nextPt.y *= 2.f;

        nextPt.x -= c_halfWin_x;
        nextPt.y -= c_halfWin_y;

        for (int k = 0; k < c_iters; ++k)
        {
            if (nextPt.x < -c_halfWin_x || nextPt.x >= cols || nextPt.y < -c_halfWin_y || nextPt.y >= rows)
            {
                if (tid == 0 && level == 0)
                    status[blockIdx.x] = 0;

                return;
            }

            float b1 = 0;
            float b2 = 0;

            for (int y = threadIdx.y, i = 0; y < c_winSize_y; y += blockDim.y, ++i)
            {
                for (int x = threadIdx.x, j = 0; x < c_winSize_x; x += blockDim.x, ++j)
                {
                    work_type I_val = I_patch[i][j];
                    work_type J_val = ToFloat<T>(J(nextPt.y + y + 0.5f, nextPt.x + x + 0.5f));

                    work_type diff = (J_val - I_val) * 32.0f;

                    accum(b1, diff * dIdx_patch[i][j]);
                    accum(b2, diff * dIdy_patch[i][j]);
                }
            }

            reduce<BLOCK_SIZE>(smem_tuple(smem1, smem2), thrust::tie(b1, b2), tid, thrust::make_tuple(plus<float>(), plus<float>()));

#if __CUDA_ARCH__ >= 300
            if (tid == 0)
            {
                smem1[0] = b1;
                smem2[0] = b2;
            }
#endif

            __syncthreads();

            b1 = smem1[0];
            b2 = smem2[0];

            float2 delta;
            delta.x = A12 * b2 - A22 * b1;
            delta.y = A12 * b1 - A11 * b2;

            nextPt.x += delta.x;
            nextPt.y += delta.y;

            if (::fabs(delta.x) < 0.01f && ::fabs(delta.y) < 0.01f)
                break;
        }

        float errval = 0;
        if (calcErr)
        {
            for (int y = threadIdx.y, i = 0; y < c_winSize_y; y += blockDim.y, ++i)
            {
                for (int x = threadIdx.x, j = 0; x < c_winSize_x; x += blockDim.x, ++j)
                {
                    work_type I_val = I_patch[i][j];
                    work_type J_val = ToFloat<T>(J(nextPt.y + y + 0.5f, nextPt.x + x + 0.5f));

                    work_type diff = J_val - I_val;

                    accum(errval, abs_(diff));
                }
            }

            reduce<BLOCK_SIZE>(smem1, errval, tid, plus<float>());
        }

        if (tid == 0)
        {
            nextPt.x += c_halfWin_x;
            nextPt.y += c_halfWin_y;

            nextPts[blockIdx.x] = nextPt;

            if (calcErr)
                err[blockIdx.x] = static_cast<float>(errval) / (::min(cn, 3)*c_winSize_x * c_winSize_y);
        }
    } // __global__ void sparseKernel_


    //template <> struct TexMaker<1, uchar>
    //{
        //static __device__ __forceinline__ float read(float x, float y)
        //{
        //    return tex2D(tex_I8U, x, y);
        //}
        //static __host__ __forceinline__ void bindTexture_(PtrStepSz<uchar>& I)
        //{
        //    bindTexture(&tex_I8U, I);
        //}
    //};

    //template <int cn, typename T> cv::cudev::Texture<T> TexMaker
    //{
    //    //static __device__ __forceinline__ float operator ()(float x, float y)
    //    //{
    //    //    return tex2D(tex_I8U, x, y);
    //    //}
    //    //static __host__ __forceinline__ void bindTexture_(PtrStepSz<typename TypeVec<T, cn>::vec_type> I)
    //    //{
    //    //    CV_UNUSED(I);
    //    //}
    //};

    //template <> struct TexMaker<1, uchar>
    //{
    //    cv::cudev::Texture<uchar> texI(cv::cudev::globPtr(src.data, src.step, src.rows, src.cols), false,
    //        hipFilterModeLinear, hipAddressModeClamp, hipReadModeNormalizedFloat);
    //};

    //// needs to return the Texture as each will have a different type
    //template <class T, int cn> struct TextureAccessor
    //{
    //    //TextureAccessor(const PtrStepSz<T,cn>& src, const int yoff_, const int xoff_) :
    //    //    tex(cv::cudev::globPtr(src.data, src.step, src.rows, src.cols), false, hipFilterModePoint, hipAddressModeClamp), yoff(yoff_), xoff(xoff_) {};

    //    TextureAccessor(PtrStepSz<typename TypeVec<uchar, 1>::vec_type> src) {

    //        cv::cudev::Texture<uchar> texI(cv::cudev::globPtr(src.data, src.step, src.rows, src.cols), false,
    //            hipFilterModeLinear, hipAddressModeClamp, hipReadModeNormalizedFloat);
    //    }

    //    cv::cudev::Texture<T> tex;
    //    //typedef T elem_type;
    //   // typedef int index_type;
    //    //int yoff;
    //   //int xoff;

    //    __device__ __forceinline__ T operator ()(int y, int x) const
    //    {
    //        return tex(y + yoff, x + xoff);
    //    }
    //};

 /*   template <int cn, typename T> struct Tex_I
    {
        static __host__ __forceinline__ void bindTexture_(PtrStepSz<typename TypeVec<T, cn>::vec_type> I)
        {
            CV_UNUSED(I);
        }
    };

    template <> struct Tex_I<1, uchar>
    {
        static __device__ __forceinline__ float read(float x, float y)
        {
            return tex2D(tex_I8U, x, y);
        }
        static __host__ __forceinline__ void bindTexture_(PtrStepSz<uchar>& I)
        {
            bindTexture(&tex_I8U, I);
        }
    };*/

    //template<class T, int cn>
    //cv::cudev::Texture <TypeVec<T, cn>::vec_type, TypeVec<float, cn>::vec_type> TextureFactory(const PtrStepSz<TypeVec<T, cn>::vec_type>& src) {
    //    return cv::cudev::Texture <typename TypeVec<T, cn>::vec_type, typename TypeVec<float, cn>::vec_type>(cv::cudev::globPtr(src.data, src.step, src.rows, src.cols),
    //        false, hipFilterModeLinear, hipAddressModeClamp, hipReadModeNormalizedFloat);
    //}

    //template <> cv::cudev::Texture <TypeVec<float, 1>::vec_type, TypeVec<float, 1>::vec_type> TextureFactory<float,1>(const PtrStepSz<TypeVec<float, 1>::vec_type>& src) {
    //    return cv::cudev::Texture <TypeVec<float, 1>::vec_type, TypeVec<float, 1>::vec_type>(cv::cudev::globPtr(src.data, src.step, src.rows, src.cols),
    //        false, hipFilterModeLinear, hipAddressModeClamp);
    //}



    //template <int cn, class T> cv::cudev::Texture <TypeVec<float, cn>::vec_type, TypeVec<float, cn>::vec_type> TextureFactory<float, cn>(const PtrStepSz<TypeVec<float, cn>::vec_type>& src) {
    //    return cv::cudev::Texture <typename TypeVec<float, cn>::vec_type, typename TypeVec<float, cn>::vec_type>(cv::cudev::globPtr(src.data, src.step, src.rows, src.cols),
    //        false, hipFilterModeLinear, hipAddressModeClamp);
    //}

    //template <int cn> cv::cudev::Texture <TypeVec<float, 1>::vec_type, TypeVec<float, 1>::vec_type> TextureFactory<float, 1>(const PtrStepSz<TypeVec<float, 1>::vec_type>& src) {
    //    return cv::cudev::Texture <typename TypeVec<float, 1>::vec_type, typename TypeVec<float, 1>::vec_type>(cv::cudev::globPtr(src.data, src.step, src.rows, src.cols),
    //        false, hipFilterModeLinear, hipAddressModeClamp);
    //}

    //template <int cn> cv::cudev::Texture <TypeVec<float, cn>::vec_type, TypeVec<float, cn>::vec_type> TextureFactory<float, 1>(const PtrStepSz<TypeVec<float, cn>::vec_type>& src) {
    //    return cv::cudev::Texture <typename TypeVec<float, cn>::vec_type, typename TypeVec<float, cn>::vec_type>(cv::cudev::globPtr(src.data, src.step, src.rows, src.cols),
    //        false, hipFilterModeLinear, hipAddressModeClamp);
    //}

    //template <class T, class R>  struct TextureAccessor
    //{
    //    //typedef TypeVec<T, cn>::vec_type elem_type;
    //    //typedef TypeVec<float, cn>::vec_type ret_type;
    //    TextureAccessor(const PtrStepSz<T>& src) :
    //        tex(cv::cudev::globPtr(src.data, src.step, src.rows, src.cols), false, hipFilterModeLinear, hipAddressModeClamp, hipReadModeNormalizedFloat) {};
    //    cv::cudev::Texture <T, R> tex;
    //    __device__ __forceinline__ R operator ()(float x, float y) const { return tex(y, x); }
    //};

// would probably need to place cn at the end?
    //template <typename T, typename R>  struct TextureAccessor
    //{
    //    typedef T elem_type;
    //    typedef R ret_type;
    //    TextureAccessor(const PtrStepSz<elem_type>& src) :
    //        tex(cv::cudev::globPtr(src.data, src.step, src.rows, src.cols), false, hipFilterModeLinear, hipAddressModeClamp) {};
    //    //TextureAccessor(const PtrStepSz<elem_type>& src) : TextureAccessor
    //    //    tex(cv::cudev::globPtr(src.data, src.step, src.rows, src.cols), false, hipFilterModeLinear, hipAddressModeClamp, hipReadModeNormalizedFloat) {};
    //    cv::cudev::Texture <elem_type, ret_type> tex;
    //    __device__ __forceinline__ ret_type operator ()(float x, float y) const { return tex(y, x); }
    //};

    //template <typename T, typename R, int cn>  struct TextureAccessor
    //{
    //    typedef TypeVec<T, cn>::vec_type elem_type;
    //    typedef TypeVec<R, cn>::vec_type ret_type;
    //    TextureAccessor(const PtrStepSz<elem_type>& src) :
    //        tex(cv::cudev::globPtr(src.data, src.step, src.rows, src.cols), false, hipFilterModeLinear, hipAddressModeClamp) {};
    //    //TextureAccessor(const PtrStepSz<elem_type>& src) : TextureAccessor
    //    //    tex(cv::cudev::globPtr(src.data, src.step, src.rows, src.cols), false, hipFilterModeLinear, hipAddressModeClamp, hipReadModeNormalizedFloat) {};
    //    cv::cudev::Texture <elem_type, ret_type> tex;
    //    __device__ __forceinline__ ret_type operator ()(float x, float y) const { return tex(y, x);}
    //};

    //template <typename T, typename R> struct TextureAccessor<T, R, 1> {
    //    typedef TypeVec<T, cn>::vec_type elem_type;
    //    typedef TypeVec<float, cn>::vec_type ret_type;
    //    TextureAccessor(const PtrStepSz<elem_type>& src) :
    //        tex(cv::cudev::globPtr(src.data, src.step, src.rows, src.cols), false, hipFilterModeLinear, hipAddressModeClamp, hipReadModeNormalizedFloat) {};
    //    cv::cudev::Texture <elem_type, ret_type> tex;
    //    __device__ __forceinline__ ret_type operator ()(float x, float y) const{return tex(y, x);}
    //};

    //template <typename T, typename R> struct TextureAccessor<T, R> {
    //    typedef TypeVec<float, cn>::vec_type elem_type;
    //    typedef TypeVec<float, cn>::vec_type ret_type;
    //    TextureAccessor(const PtrStepSz<elem_type>& src) :
    //        tex(cv::cudev::globPtr(src.data, src.step, src.rows, src.cols), false, hipFilterModeLinear, hipAddressModeClamp) {};
    //    cv::cudev::Texture <elem_type, ret_type> tex;
    //    __device__ __forceinline__ ret_type operator ()(float x, float y) const { return tex(y, x); }
    //};



// Need a template which takes in cn if cn == 1 passes through the type if not creates TypeVec<T, cn>::vec_type




//cv::cuda::device::TypeVec

    //template <> struct TextureAccessor<1, float> {
    //    TextureAccessor(const PtrStepSz<typename TypeVec<float, 1>::vec_type>& src) :
    //        tex(cv::cudev::globPtr(src.data, src.step, src.rows, src.cols), false, hipFilterModeLinear, hipAddressModeClamp) {};
    //    cv::cudev::Texture <TypeVec<float, 1>::vec_type, TypeVec<float, 1>::vec_type> tex;
    //    __device__ __forceinline__ TypeVec<float, 1>::vec_type operator ()(float x, float y) const {
    //        printf("x: %f, y: %f, ", x, y);
    //        float tmp = tex(y, x);
    //        printf("val: %d\n", tmp);
    //        return tex(y, x);
    //    }
    //};
    //{

        //static cv::cudev::Texture <typename TypeVec<float, 1>::vec_type, typename TypeVec<float, 1>::vec_type> Test(const PtrStepSz<typename TypeVec<float, 1>::vec_type>& src) {
        //    return cv::cudev::Texture <typename TypeVec<float, 1>::vec_type, typename TypeVec<float, 1>::vec_type>(
        //        cv::cudev::globPtr(src.data, src.step, src.rows, src.cols), false, hipFilterModeLinear, hipAddressModeClamp);
        //}
    //};

    //template <> struct TextureAccessor<float, 4>
    //{
    //    static cv::cudev::Texture <typename TypeVec<float, 4>::vec_type, typename TypeVec<float, 4>::vec_type> Test(const PtrStepSz<typename TypeVec<float, 4>::vec_type>& src) {
    //        return cv::cudev::Texture <typename TypeVec<float, 4>::vec_type, typename TypeVec<float, 4>::vec_type>(
    //            cv::cudev::globPtr(src.data, src.step, src.rows, src.cols), false, hipFilterModeLinear, hipAddressModeClamp);
    //    }
    //};

// no offset, specialized only for this class where the return type is always float - see note.
// Could name it to FloatTextureAccessor

//template <int cn, typename T>  struct TextureAccessor
//{
//    typedef typename TypeVec<T, cn>::vec_type elem_type;
//    typedef typename TypeVec<float, cn>::vec_type ret_type;
//    TextureAccessor(const PtrStepSz<elem_type>& src) :
//        tex(src, false, hipFilterModeLinear, hipAddressModeClamp, hipReadModeNormalizedFloat)
//    {
//        texPtr = tex;
//    };
//    cv::cudev::Texture <elem_type, ret_type> tex;
//    cv::cudev::TexturePtr<elem_type, ret_type> texPtr;
//    __device__ __forceinline__ ret_type operator ()(float x, float y) const { return texPtr(y, x); }
//};

// FilterModeLinear only works with floating point return tpe
template<int cn, class T> class TextureLinear : public cv::cudev::Texture<TypeVec<T, cn>::vec_type, TypeVec<float, cn>::vec_type> {
public:
    typedef typename TypeVec<T, cn>::vec_type elem_type;
    typedef typename TypeVec<float, cn>::vec_type ret_type;
    //__host__ TextureLinear(const int rows, const int cols, T* data, const size_t step, const bool normalizedCoords = false,const hipTextureAddressMode addressMode = hipAddressModeClamp) :
    //    cv::cudev::Texture<elem_type, ret_type>(rows,cols,data,step, normalizedCoords, hipFilterModeLinear, addressMode, hipReadModeNormalizedFloat)
    //{
    //}

    __host__ TextureLinear(PtrStepSz<elem_type> src, const bool normalizedCoords = false, const hipTextureAddressMode addressMode = hipAddressModeClamp) :
        cv::cudev::Texture<elem_type, ret_type>(src, normalizedCoords, hipFilterModeLinear, addressMode, hipReadModeNormalizedFloat)
    {
    }
};

//template<class T> class TextureLinear : public cv::cudev::Texture<T, float4> {
//public:
//    __host__ TextureLinear(const int rows, const int cols, T* data, const size_t step, const bool normalizedCoords = false, const hipTextureAddressMode addressMode = hipAddressModeClamp) :
//        cv::cudev::Texture<T, float>(rows, cols, data, step, normalizedCoords, hipFilterModeLinear, addressMode, hipReadModeNormalizedFloat)
//    {
//    }
//
//    __host__ TextureLinear(PtrStepSz<T> src, const bool normalizedCoords = false, const hipTextureAddressMode addressMode = hipAddressModeClamp) :
//        cv::cudev::Texture<T, float>(src, normalizedCoords, hipFilterModeLinear, addressMode, hipReadModeNormalizedFloat)
//    {
//    }
//};

// hipReadModeNormalizedFloat doesn't work if data is a float
template<int cn> class TextureLinear<cn,float> : public cv::cudev::Texture<TypeVec<float, cn>::vec_type, TypeVec<float, cn>::vec_type>
{
public:
    //__host__ TextureLinear(const int rows, const int cols, float* data, const size_t step, const bool normalizedCoords = false, const hipTextureAddressMode addressMode = hipAddressModeClamp) :
    //    cv::cudev::Texture <float, float>(rows, cols, data, step, normalizedCoords, hipFilterModeLinear, addressMode, hipReadModeElementType)
    //{
    //}
    typedef typename TypeVec<float, cn>::vec_type float_type;
    __host__ TextureLinear(PtrStepSz<float_type> src, const bool normalizedCoords = false, const hipTextureAddressMode addressMode = hipAddressModeClamp) :
        cv::cudev::Texture <float_type, float_type>(src, normalizedCoords, hipFilterModeLinear, addressMode, hipReadModeElementType)
    {
    }
};

//template<> class TextureLinear<float4> : public cv::cudev::Texture<float4, float4>
//{
//public:
//    __host__ TextureLinear(const int rows, const int cols, float* data, const size_t step, const bool normalizedCoords = false, const hipTextureAddressMode addressMode = hipAddressModeClamp) :
//        cv::cudev::Texture <float, float>(rows, cols, data, step, normalizedCoords, hipFilterModeLinear, addressMode, hipReadModeElementType)
//    {
//    }
//    __host__ TextureLinear(PtrStepSz<float> src, const bool normalizedCoords = false, const hipTextureAddressMode addressMode = hipAddressModeClamp) :
//        cv::cudev::Texture <float, float>(src, normalizedCoords, hipFilterModeLinear, addressMode, hipReadModeElementType)
//    {
//    }
//};



//template <int cn, typename T> struct TextureAccessor<cn, T> {
//    typedef TypeVec<T, cn>::vec_type elem_type;
//    typedef TypeVec<float, cn>::vec_type ret_type;
//    TextureAccessor(const PtrStepSz<elem_type>& src) :
//        tex(cv::cudev::globPtr(src.data, src.step, src.rows, src.cols), false, hipFilterModeLinear, hipAddressModeClamp, hipReadModeNormalizedFloat) {};
//    cv::cudev::Texture <elem_type, ret_type> tex;
//    __device__ __forceinline__ ret_type operator ()(float x, float y) const { return tex(y, x); }
//};

// hipReadModeNormalizedFloat throws an error when trying to promote float values
// Nvidia Docs for hipTextureDesc::readMode: Note that this applies only to 8 - bit and 16 - bit integer formats. 32 - bit integer format would not be promoted,
// regardless of whether or not this hipTextureDesc::readMode is set hipReadModeNormalizedFloat is specified.
//template <int cn> struct TextureAccessor<cn, float> {
//    typedef typename TypeVec<float, cn>::vec_type float_type;
//    TextureAccessor(const PtrStepSz<float_type>& src) :
//        tex(src, false, hipFilterModeLinear, hipAddressModeClamp, hipReadModeElementType)
//    {
//        texPtr = tex;
//    };
//    cv::cudev::Texture <float_type, float_type> tex;
//    cv::cudev::TexturePtr<float_type, float_type> texPtr;
//    __device__ __forceinline__ float_type operator ()(float x, float y) const { return texPtr(y, x); }
//};


    template <int cn, int PATCH_X, int PATCH_Y, typename T> class sparse_caller
    {
    public:
        static void call(PtrStepSz<typename TypeVec<T, cn>::vec_type> I, PtrStepSz<typename TypeVec<T, cn>::vec_type> J, int rows, int cols, const float2* prevPts, float2* nextPts, uchar* status, float* err, int ptcount,
            int level, dim3 block, hipStream_t stream)
        {
            dim3 grid(ptcount);
            //CV_UNUSED(I);
            //CV_UNUSED(J);

            //typedef typename TypeVec<float, cn>::vec_type work_type;
            //work_type tmp;

            //float4 tmp1;
            //tmp1.x = 3;

            //cv::cudev::Texture<T> texI(cv::cudev::globPtr((T*)(I.data), I.step, I.rows, I.cols), false, hipFilterModeLinear, hipAddressModeClamp, hipReadModeNormalizedFloat);
            //cv::cudev::Texture<T> texJ(cv::cudev::globPtr((T*)(J.data), J.step, J.rows, J.cols), false, hipFilterModeLinear, hipAddressModeClamp, hipReadModeNormalizedFloat);

            //cv::cudev::Texture<typename TypeVec<T, cn>::vec_type, typename TypeVec<float, cn>::vec_type> texI(cv::cudev::globPtr((I.data), I.step, I.rows, I.cols), false, hipFilterModeLinear, hipAddressModeClamp, hipReadModeNormalizedFloat);
            //TextureAccessor<cn, T> texAccess(I);
            //cv::cudev::Texture<typename TypeVec<T, cn>::vec_type, typename TypeVec<float, cn>::vec_type> texTst = TextureAccessor<T,cn>::Test(I);

            //cv::cudev::Texture<typename TypeVec<T, cn>::vec_type, typename TypeVec<float, cn>::vec_type> texI = TextureFactory<cn,T>(I);
            //cv::cudev::Texture<typename TypeVec<T, cn>::vec_type, typename TypeVec<float, cn>::vec_type> texJ = TextureFactory<cn, T>(J);
            //auto texI = TextureFactory<T,cn>(I);
            //auto texJ = TextureFactory<T,cn>(J);
            typedef typename TypeVec<T, cn>::vec_type dType;
            typedef typename TypeVec<float, cn>::vec_type rType;
            //auto texI = TextureAccessor<cn, T>(I);

            //auto texJ = TextureAccessor<cn, T>(J);

            //cv::cudev::Texture<dType, rType> texI(I, false, hipFilterModeLinear);
            //cv::cudev::Texture<dType, rType> texJ(J, false, hipFilterModeLinear);

            TextureLinear<cn,T> texI(I);
            TextureLinear<cn,T> texJ(J);

            //cv::cudev::TexturePtr<dType, rType> texPtrI = texI;



            //cv::cudev::Texture<typename TypeVec<T, cn>::vec_type, typename TypeVec<float, cn>::vec_type> texJ(cv::cudev::globPtr((J.data), J.step, J.rows, J.cols), false, hipFilterModeLinear, hipAddressModeClamp);
            if (level == 0 && err)
                sparseKernel<cn, PATCH_X, PATCH_Y, true, T, cv::cudev::TexturePtr<dType,rType>> << <grid, block, 0, stream >> > (texI, texJ, prevPts, nextPts, status, err, level, rows, cols);
            else
                sparseKernel<cn, PATCH_X, PATCH_Y, false, T, cv::cudev::TexturePtr<dType, rType>> << <grid, block, 0, stream >> > (texI, texJ, prevPts, nextPts, status, err, level, rows, cols);

            //if (level == 0 && err)
            //    sparseKernel<cn, PATCH_X, PATCH_Y, true, T> <<<grid, block, 0, stream >>>(prevPts, nextPts, status, err, level, rows, cols);
            //else
            //    sparseKernel<cn, PATCH_X, PATCH_Y, false, T> <<<grid, block, 0, stream >>>(prevPts, nextPts, status, err, level, rows, cols);

            cudaSafeCall(hipGetLastError());

            if (stream == 0)
                cudaSafeCall(hipDeviceSynchronize());
        }
    };
    // Specialization to use non texture path because for some reason the texture path keeps failing accuracy tests
    template<int PATCH_X, int PATCH_Y> class sparse_caller<1, PATCH_X, PATCH_Y, unsigned short>
    {
    public:
        typedef typename TypeVec<unsigned short, 1>::vec_type work_type;
        typedef PtrStepSz<work_type> Ptr2D;
        typedef BrdConstant<work_type> BrdType;
        typedef BorderReader<Ptr2D, BrdType> Reader;
        typedef LinearFilter<Reader> Filter;
        static void call(Ptr2D I, Ptr2D J, int rows, int cols, const float2* prevPts, float2* nextPts, uchar* status, float* err, int ptcount,
            int level, dim3 block, hipStream_t stream)
        {
            dim3 grid(ptcount);
            if (level == 0 && err)
            {
                sparseKernel_<PATCH_X, PATCH_Y, true, 1, unsigned short> <<<grid, block, 0, stream >>>(
                    Filter(Reader(I, BrdType(rows, cols))),
                    Filter(Reader(J, BrdType(rows, cols))),
                    prevPts, nextPts, status, err, level, rows, cols);
            }
            else
            {
                sparseKernel_<PATCH_X, PATCH_Y, false, 1, unsigned short> <<<grid, block, 0, stream >>>(
                    Filter(Reader(I, BrdType(rows, cols))),
                    Filter(Reader(J, BrdType(rows, cols))),
                    prevPts, nextPts, status, err, level, rows, cols);
            }
            cudaSafeCall(hipGetLastError());

            if (stream == 0)
                cudaSafeCall(hipDeviceSynchronize());
        }
    };
    // Specialization for int because the texture path keeps failing
    template<int PATCH_X, int PATCH_Y> class sparse_caller<1, PATCH_X, PATCH_Y, int>
    {
    public:
        typedef typename TypeVec<int, 1>::vec_type work_type;
        typedef PtrStepSz<work_type> Ptr2D;
        typedef BrdConstant<work_type> BrdType;
        typedef BorderReader<Ptr2D, BrdType> Reader;
        typedef LinearFilter<Reader> Filter;
        static void call(Ptr2D I, Ptr2D J, int rows, int cols, const float2* prevPts, float2* nextPts, uchar* status, float* err, int ptcount,
            int level, dim3 block, hipStream_t stream)
        {
            dim3 grid(ptcount);
            if (level == 0 && err)
            {
                sparseKernel_<PATCH_X, PATCH_Y, true, 1, int> <<<grid, block, 0, stream >>>(
                    Filter(Reader(I, BrdType(rows, cols))),
                    Filter(Reader(J, BrdType(rows, cols))),
                    prevPts, nextPts, status, err, level, rows, cols);
            }
            else
            {
                sparseKernel_<PATCH_X, PATCH_Y, false, 1, int> <<<grid, block, 0, stream >>>(
                    Filter(Reader(I, BrdType(rows, cols))),
                    Filter(Reader(J, BrdType(rows, cols))),
                    prevPts, nextPts, status, err, level, rows, cols);
            }
            cudaSafeCall(hipGetLastError());

            if (stream == 0)
                cudaSafeCall(hipDeviceSynchronize());
        }
    };
    template<int PATCH_X, int PATCH_Y> class sparse_caller<4, PATCH_X, PATCH_Y, int>
    {
    public:
        typedef typename TypeVec<int, 4>::vec_type work_type;
        typedef PtrStepSz<work_type> Ptr2D;
        typedef BrdConstant<work_type> BrdType;
        typedef BorderReader<Ptr2D, BrdType> Reader;
        typedef LinearFilter<Reader> Filter;
        static void call(Ptr2D I, Ptr2D J, int rows, int cols, const float2* prevPts, float2* nextPts, uchar* status, float* err, int ptcount,
            int level, dim3 block, hipStream_t stream)
        {
            dim3 grid(ptcount);
            if (level == 0 && err)
            {
                sparseKernel_<PATCH_X, PATCH_Y, true, 4, int> <<<grid, block, 0, stream >>>(
                    Filter(Reader(I, BrdType(rows, cols))),
                    Filter(Reader(J, BrdType(rows, cols))),
                    prevPts, nextPts, status, err, level, rows, cols);
            }
            else
            {
                sparseKernel_<PATCH_X, PATCH_Y, false, 4, int> <<<grid, block, 0, stream >>>(
                    Filter(Reader(I, BrdType(rows, cols))),
                    Filter(Reader(J, BrdType(rows, cols))),
                    prevPts, nextPts, status, err, level, rows, cols);
            }
            cudaSafeCall(hipGetLastError());

            if (stream == 0)
                cudaSafeCall(hipDeviceSynchronize());
        }
    };
    using namespace cv::cuda::device;
    template <int PATCH_X, int PATCH_Y, typename T> class sparse_caller<3, PATCH_X, PATCH_Y, T>
    {
    public:
        typedef typename TypeVec<T, 3>::vec_type work_type;
        typedef PtrStepSz<work_type> Ptr2D;
        typedef BrdConstant<work_type> BrdType;
        typedef BorderReader<Ptr2D, BrdType> Reader;
        typedef LinearFilter<Reader> Filter;
        static void call(Ptr2D I, Ptr2D J, int rows, int cols, const float2* prevPts, float2* nextPts, uchar* status, float* err, int ptcount,
            int level, dim3 block, hipStream_t stream)
        {
            dim3 grid(ptcount);
            if (level == 0 && err)
            {
                sparseKernel_<PATCH_X, PATCH_Y, true, 3, T> <<<grid, block, 0, stream >>>(
                    Filter(Reader(I, BrdType(rows, cols))),
                    Filter(Reader(J, BrdType(rows, cols))),
                    prevPts, nextPts, status, err, level, rows, cols);
            }
            else
            {
                sparseKernel_<PATCH_X, PATCH_Y, false, 3, T> <<<grid, block, 0, stream >>>(
                    Filter(Reader(I, BrdType(rows, cols))),
                    Filter(Reader(J, BrdType(rows, cols))),
                    prevPts, nextPts, status, err, level, rows, cols);
            }
            cudaSafeCall(hipGetLastError());

            if (stream == 0)
                cudaSafeCall(hipDeviceSynchronize());
        }
    };


    template <bool calcErr, class Ptr2D>
    __global__ void denseKernel(const Ptr2D texI, const Ptr2D texJ, PtrStepf u, PtrStepf v, const PtrStepf prevU, const PtrStepf prevV, PtrStepf err, const int rows, const int cols)
    {
        extern __shared__ int smem[];

        const int patchWidth  = blockDim.x + 2 * c_halfWin_x;
        const int patchHeight = blockDim.y + 2 * c_halfWin_y;

        int* I_patch = smem;
        int* dIdx_patch = I_patch + patchWidth * patchHeight;
        int* dIdy_patch = dIdx_patch + patchWidth * patchHeight;

        const int xBase = blockIdx.x * blockDim.x;
        const int yBase = blockIdx.y * blockDim.y;

        for (int i = threadIdx.y; i < patchHeight; i += blockDim.y)
        {
            for (int j = threadIdx.x; j < patchWidth; j += blockDim.x)
            {
                float x = xBase - c_halfWin_x + j + 0.5f;
                float y = yBase - c_halfWin_y + i + 0.5f;

                I_patch[i * patchWidth + j] = texI(y, x);

                // Scharr Deriv

                //dIdx_patch[i * patchWidth + j] = 3 * tex2D(tex_If, x+1, y-1) + 10 * tex2D(tex_If, x+1, y) + 3 * tex2D(tex_If, x+1, y+1) -
                //                                (3 * tex2D(tex_If, x-1, y-1) + 10 * tex2D(tex_If, x-1, y) + 3 * tex2D(tex_If, x-1, y+1));

                //dIdy_patch[i * patchWidth + j] = 3 * tex2D(tex_If, x-1, y+1) + 10 * tex2D(tex_If, x, y+1) + 3 * tex2D(tex_If, x+1, y+1) -
                //                                (3 * tex2D(tex_If, x-1, y-1) + 10 * tex2D(tex_If, x, y-1) + 3 * tex2D(tex_If, x+1, y-1));

                dIdx_patch[i * patchWidth + j] = 3 * texI(y - 1, x + 1) + 10 * texI(y, x + 1) + 3 * texI(y + 1, x + 1) -
                    (3 * texI(y - 1, x - 1) + 10 * texI(y, x - 1) + 3 * texI(y + 1, x - 1));

                dIdy_patch[i * patchWidth + j] = 3 * texI(y + 1, x - 1) + 10 * texI(y + 1,x) + 3 * texI(y+ 1, x + 1) -
                    (3 * texI(y - 1, x - 1) + 10 * texI(y - 1,x) + 3 * texI(y - 1, x + 1));
            }
        }

        __syncthreads();

        const int x = xBase + threadIdx.x;
        const int y = yBase + threadIdx.y;

        if (x >= cols || y >= rows)
            return;


        int A11i = 0;
        int A12i = 0;
        int A22i = 0;

        for (int i = 0; i < c_winSize_y; ++i)
        {
            for (int j = 0; j < c_winSize_x; ++j)
            {
                int dIdx = dIdx_patch[(threadIdx.y + i) * patchWidth + (threadIdx.x + j)];
                int dIdy = dIdy_patch[(threadIdx.y + i) * patchWidth + (threadIdx.x + j)];

                A11i += dIdx * dIdx;
                A12i += dIdx * dIdy;
                A22i += dIdy * dIdy;
            }
        }

        float A11 = A11i;
        float A12 = A12i;
        float A22 = A22i;

        float D = A11 * A22 - A12 * A12;

        if (D < numeric_limits<float>::epsilon())
        {
            if (calcErr)
                err(y, x) = numeric_limits<float>::max();
            return;
        }

        D = 1.f / D;

        A11 *= D;
        A12 *= D;
        A22 *= D;

        float2 nextPt;
        nextPt.x = x + prevU(y/2, x/2) * 2.0f;
        nextPt.y = y + prevV(y/2, x/2) * 2.0f;

        for (int k = 0; k < c_iters; ++k)
        {
            if (nextPt.x < 0 || nextPt.x >= cols || nextPt.y < 0 || nextPt.y >= rows)
            {
                if (calcErr)
                    err(y, x) = numeric_limits<float>::max();

                return;
            }

            int b1 = 0;
            int b2 = 0;

            for (int i = 0; i < c_winSize_y; ++i)
            {
                for (int j = 0; j < c_winSize_x; ++j)
                {
                    int I = I_patch[(threadIdx.y + i) * patchWidth + threadIdx.x + j];
                    //int J = tex2D(tex_Jf, nextPt.x - c_halfWin_x + j + 0.5f, nextPt.y - c_halfWin_y + i + 0.5f);
                    int J = texJ(nextPt.y - c_halfWin_y + i + 0.5f, nextPt.x - c_halfWin_x + j + 0.5f);

                    int diff = (J - I) * 32;

                    int dIdx = dIdx_patch[(threadIdx.y + i) * patchWidth + (threadIdx.x + j)];
                    int dIdy = dIdy_patch[(threadIdx.y + i) * patchWidth + (threadIdx.x + j)];

                    b1 += diff * dIdx;
                    b2 += diff * dIdy;
                }
            }


            float2 delta;
            delta.x = A12 * b2 - A22 * b1;
            delta.y = A12 * b1 - A11 * b2;

            nextPt.x += delta.x;
            nextPt.y += delta.y;

            if (::fabs(delta.x) < 0.01f && ::fabs(delta.y) < 0.01f)
                break;
        }

        u(y, x) = nextPt.x - x;
        v(y, x) = nextPt.y - y;

        if (calcErr)
        {
            int errval = 0;

            for (int i = 0; i < c_winSize_y; ++i)
            {
                for (int j = 0; j < c_winSize_x; ++j)
                {
                    int I = I_patch[(threadIdx.y + i) * patchWidth + threadIdx.x + j];
                    //int J = tex2D(tex_Jf, nextPt.x - c_halfWin_x + j + 0.5f, nextPt.y - c_halfWin_y + i + 0.5f);
                    int J = texJ(nextPt.y - c_halfWin_y + i + 0.5f, nextPt.x - c_halfWin_x + j + 0.5f);

                    errval += ::abs(J - I);
                }
            }

            err(y, x) = static_cast<float>(errval) / (c_winSize_x * c_winSize_y);
        }
    }

    void loadWinSize(int* winSize, int* halfWinSize, hipStream_t stream)
    {
        cudaSafeCall( hipMemcpyToSymbolAsync(HIP_SYMBOL(c_winSize_x), winSize, sizeof(int), 0, hipMemcpyHostToDevice, stream) );
        cudaSafeCall( hipMemcpyToSymbolAsync(HIP_SYMBOL(c_winSize_y), winSize + 1, sizeof(int), 0, hipMemcpyHostToDevice, stream) );

        cudaSafeCall( hipMemcpyToSymbolAsync(HIP_SYMBOL(c_halfWin_x), halfWinSize, sizeof(int), 0, hipMemcpyHostToDevice, stream) );
        cudaSafeCall( hipMemcpyToSymbolAsync(HIP_SYMBOL(c_halfWin_y), halfWinSize + 1, sizeof(int), 0, hipMemcpyHostToDevice, stream) );
    }

    void loadIters(int* iters, hipStream_t stream)
    {
        cudaSafeCall( hipMemcpyToSymbolAsync(HIP_SYMBOL(c_iters), iters, sizeof(int), 0, hipMemcpyHostToDevice, stream) );
    }

    void loadConstants(int2 winSize_, int iters_, hipStream_t stream)
    {
        static int2 winSize = make_int2(0,0);
        if(winSize.x != winSize_.x || winSize.y != winSize_.y)
        {
            winSize = winSize_;
            cudaSafeCall( hipMemcpyToSymbolAsync(HIP_SYMBOL(c_winSize_x), &winSize.x, sizeof(int), 0, hipMemcpyHostToDevice, stream) );
            cudaSafeCall( hipMemcpyToSymbolAsync(HIP_SYMBOL(c_winSize_y), &winSize.y, sizeof(int), 0, hipMemcpyHostToDevice, stream) );
        }

        static int2 halfWin = make_int2(0,0);
        int2 half = make_int2((winSize.x - 1) / 2, (winSize.y - 1) / 2);
        if(halfWin.x != half.x || halfWin.y != half.y)
        {
            halfWin = half;
            cudaSafeCall( hipMemcpyToSymbolAsync(HIP_SYMBOL(c_halfWin_x), &halfWin.x, sizeof(int), 0, hipMemcpyHostToDevice, stream) );
            cudaSafeCall( hipMemcpyToSymbolAsync(HIP_SYMBOL(c_halfWin_y), &halfWin.y, sizeof(int), 0, hipMemcpyHostToDevice, stream) );
        }

        static int iters = 0;
        if(iters != iters_)
        {
            iters = iters_;
            cudaSafeCall( hipMemcpyToSymbolAsync(HIP_SYMBOL(c_iters), &iters, sizeof(int), 0, hipMemcpyHostToDevice, stream) );
        }
    }

    template<typename T, int cn> struct pyrLK_caller
    {
        static void sparse(PtrStepSz<typename TypeVec<T, cn>::vec_type> I, PtrStepSz<typename TypeVec<T, cn>::vec_type> J, const float2* prevPts, float2* nextPts, uchar* status, float* err, int ptcount,
            int level, dim3 block, dim3 patch, hipStream_t stream)
        {
            typedef void(*func_t)(PtrStepSz<typename TypeVec<T, cn>::vec_type> I, PtrStepSz<typename TypeVec<T, cn>::vec_type> J,
                int rows, int cols, const float2* prevPts, float2* nextPts, uchar* status, float* err, int ptcount,
                int level, dim3 block, hipStream_t stream);

            static const func_t funcs[5][5] =
            {
                { sparse_caller<cn, 1, 1,T>::call, sparse_caller<cn, 2, 1,T>::call, sparse_caller<cn, 3, 1,T>::call, sparse_caller<cn, 4, 1,T>::call, sparse_caller<cn, 5, 1,T>::call },
                { sparse_caller<cn, 1, 2,T>::call, sparse_caller<cn, 2, 2,T>::call, sparse_caller<cn, 3, 2,T>::call, sparse_caller<cn, 4, 2,T>::call, sparse_caller<cn, 5, 2,T>::call },
                { sparse_caller<cn, 1, 3,T>::call, sparse_caller<cn, 2, 3,T>::call, sparse_caller<cn, 3, 3,T>::call, sparse_caller<cn, 4, 3,T>::call, sparse_caller<cn, 5, 3,T>::call },
                { sparse_caller<cn, 1, 4,T>::call, sparse_caller<cn, 2, 4,T>::call, sparse_caller<cn, 3, 4,T>::call, sparse_caller<cn, 4, 4,T>::call, sparse_caller<cn, 5, 4,T>::call },
                { sparse_caller<cn, 1, 5,T>::call, sparse_caller<cn, 2, 5,T>::call, sparse_caller<cn, 3, 5,T>::call, sparse_caller<cn, 4, 5,T>::call, sparse_caller<cn, 5, 5,T>::call }
            };

            //Tex_I<cn, T>::bindTexture_(I);
            //Tex_J<cn, T>::bindTexture_(J);

            funcs[patch.y - 1][patch.x - 1](I, J, I.rows, I.cols, prevPts, nextPts, status, err, ptcount,
                level, block, stream);
        }
        static void dense(PtrStepSz<T> I, PtrStepSz<T> J, PtrStepSzf u, PtrStepSzf v, PtrStepSzf prevU, PtrStepSzf prevV, PtrStepSzf err, int2 winSize, hipStream_t stream)
        {
            dim3 block(16, 16);
            dim3 grid(divUp(I.cols, block.x), divUp(I.rows, block.y));
            //Tex_I<1, T>::bindTexture_(I);
            //Tex_J<1, T>::bindTexture_(J);
            //auto texI = TextureAccessor<1, T>(I);
            //auto texJ = TextureAccessor<1, T>(J);
            TextureLinear<1, T> texI(I);
            TextureLinear<1, T> texJ(J);
            int2 halfWin = make_int2((winSize.x - 1) / 2, (winSize.y - 1) / 2);
            const int patchWidth = block.x + 2 * halfWin.x;
            const int patchHeight = block.y + 2 * halfWin.y;
            size_t smem_size = 3 * patchWidth * patchHeight * sizeof(int);

            if (err.data)
            {
                denseKernel<true, cv::cudev::TexturePtr<T,float>> << <grid, block, smem_size, stream >> >(texI, texJ, u, v, prevU, prevV, err, I.rows, I.cols);
                cudaSafeCall(hipGetLastError());
            }
            else
            {
                denseKernel<false, cv::cudev::TexturePtr<T, float>> << <grid, block, smem_size, stream >> >(texI, texJ, u, v, prevU, prevV, PtrStepf(), I.rows, I.cols);
                cudaSafeCall(hipGetLastError());
            }

            if (stream == 0)
                cudaSafeCall(hipDeviceSynchronize());
        }
    };

    template class pyrLK_caller<unsigned char,1>;
    template class pyrLK_caller<unsigned short,1>;
    template class pyrLK_caller<int,1>;
    template class pyrLK_caller<float,1>;

    template class pyrLK_caller<unsigned char, 3>;
    template class pyrLK_caller<unsigned short, 3>;
    template class pyrLK_caller<int, 3>;
    template class pyrLK_caller<float, 3>;

    template class pyrLK_caller<unsigned char, 4>;
    template class pyrLK_caller<unsigned short, 4>;
    template class pyrLK_caller<int, 4>;
    template class pyrLK_caller<float, 4>;
}

#endif /* CUDA_DISABLER */
